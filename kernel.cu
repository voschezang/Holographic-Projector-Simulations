#include "hip/hip_runtime.h"
#ifndef KERNEL
#define KERNEL

#include <assert.h>
// #include <math.h>
#include <stdio.h>
#include <time.h>
// #include <complex.h>
#include <hip/hip_complex.h>

#include "macros.h"

#define cu(result) { cudaCheck((result), __FILE__, __LINE__); }

inline
hipError_t cudaCheck(hipError_t result, const char *file, int line)
{
  // check for cuda errors
// #ifdef DEBUG
  if (result != hipSuccess) {
    fprintf(stderr, "[%s:%d] CUDA Runtime Error: %s\n", file, line, hipGetErrorString(result));
    // assert(result == hipSuccess);
    exit(result);
  }
// #endif
  return result;
}

__device__ void cuCheck(hipDoubleComplex  z) {
  double a = hipCreal(z), b = hipCimag(z);
  if (isnan(a)) printf("cu found nan re\n");
  if (isinf(a)) printf("cu found inf re\n");
  if (isnan(b)) printf("cu found nan I\n");
  if (isinf(b)) printf("cu found inf I\n");
}

inline
__host__ __device__ double angle(hipDoubleComplex  z) {
  return atan2(hipCreal(z), hipCimag(z));
}

inline __device__ hipDoubleComplex polar(double a, double phi) {
  // Convert polar coordinates (a,phi) to complex number a * e^(phi I)
  hipDoubleComplex res;
  sincos(phi, &res.x, &res.y);
  // return hipCmul(make_hipDoubleComplex(a, 0), res);
  return make_hipDoubleComplex(a * res.x, a * res.y);
}


__global__ void kernel_zero(WTYPE_cuda *x, size_t n) {
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < n; i += stride)
    x[i] = ZERO;
}

// TODO consider non-complex types (double real, double imag)
// and check computational cost
inline __device__ WTYPE_cuda superposition_single(const size_t i, const size_t j,
                        const WTYPE_cuda *x, const STYPE *u, STYPE *v,
                        const char direction) {
  // TODO unpack input to u1,u2,3 v1,v2,v3?
  // TODO consider unguarded functions, intrinsic functions
#ifdef DEBUG
  assert(direction == -1 || direction == 1);
#endif

  size_t
    n = i * DIMS,
    m = j * DIMS; // TODO use struct?
  // TODO use softeningSquared?
  // TODO check coalesing
  double
    distance = norm3d(v[m] - u[n], v[m+1] - u[n+1], v[m+2] - u[n+2]),
    amp = hipCabs(x[i]),
    phase = angle(x[i]);

#ifdef DEBUG
  if (distance == 0) { printf("ERROR: distance must be nonzero\n"); asm("trap;"); }
  // if (amp > 0) printf(">0 \ti: %i, abs: %0.4f, dis: %0.3f\n", i, amp, distance);
  // // TODO check overflows
  if (isnan(amp)) printf("found nan\n");
  if (isinf(amp)) printf("found inf\n");
  if (isnan(distance)) printf("found nan\n");
  if (isinf(distance)) printf("found inf\n");
  // if (amp > 0) printf("amp = %0.5f > 0\n", amp);
  // if (distance > 0) printf("dis: %0.4f\n\n", distance);
  hipDoubleComplex res = polar(amp, phase);
  if (amp > 0) assert(hipCabs(res) > 0);
#endif

  // TODO __ddiv_rd, __dmul_ru
  return polar(amp / distance, phase - distance * direction * TWO_PI_OVER_LAMBDA);
}

// TODO optimize memory / prevent Shared memory bank conflicts for x,u arrays
// TODO use __restrict__, const
__global__ void kernel3(WTYPE_cuda *x, STYPE *u, double *y, STYPE *v,
                        const char direction)
{
  /** First compute local sum, then do nested aggregation
   *
   * out[BATCH_SIZE * blockDim] = array with output per block
   * v[BATCH_SIZE * DIM] = locations of y-datapoints
   */
  //
  // TODO use shared mem for u-data
  __shared__ WTYPE_cuda tmp[THREADS_PER_BLOCK * BATCH_SIZE];
#ifdef CACHE_BATCH
  // cache v[batch] because it is read by every thread
  // v_cached is constant and equal for each block
  __shared__ STYPE v_cached[BATCH_SIZE * DIMS];
  // use strides when THREADS_PER_BLOCK < BATCH_SIZE * DIMS
  for (unsigned int i = threadIdx.x; i < BATCH_SIZE * DIMS; i+=THREADS_PER_BLOCK)
    v_cached[i] = v[i];

  __syncthreads();

  // if (threadIdx.x == 0 && blockIdx.x == 0 && i_batch == 0) {
  //   printf("\n v: \t\t");
  //   for (unsigned int i = 0; i < 2 * DIMS; i+=1)
  //     printf("%8f ", v[i]);

  //   printf("\n v_cached: \t");
  //   for (unsigned int i = 0; i < 2 * DIMS; i+=1)
  //     printf("%8f ", v_cached[i]);

  //   printf("\n");
  // }
#endif
  // TODO use cuda.y-stride? - note the double for loop - how much memory fits in an SM?
  // TODO switch y-loop and x-loop and let sum : [BATCH_SIZE]? assuming y-batch is in local memory
  // printf("idx %i -", threadIdx.x);
  {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    // size_t j;
    WTYPE_cuda sum;

    // for each y-datapoint in current batch
    // TODO test performance diff when switching inner/outer loop and with um cache
    // TODO change cache size and find new optimal batch size w/ um cache
    for(unsigned int m = 0; m < BATCH_SIZE; ++m) {
// #ifndef CACHE_BATCH
//       const size_t j = m + i_batch * BATCH_SIZE;
// #endif
      sum = ZERO;
      // Usage of stride allows <<<1,1>>> kernel invocation
      for (size_t i = idx; i < N; i += stride) {
// #ifdef CACHE_BATCH
        // assert(v[(m + i_batch * BATCH_SIZE) * DIMS] == v_cached[m * DIMS]);
        sum = hipCadd(superposition_single(i, m, x, u, v_cached, direction), sum);
// #else
//         sum = hipCadd(superposition_single(i, j, x, u, v, direction), sum);
// #endif
        // TODO do this in separate func
        //TODO err: i_batch does not depend on x
      }
      // assume threadIdx.x is a runtime constant
      if (direction == -1 && threadIdx.x == 0 && idx == 0) {
        // add single far away light source, with arbitrary (but constant) phase
        // TODO this causes a strange offset in z
        sum = hipCadd(polar(1, 0.4912), sum);
      }
      sum = hipCadd(polar(1, 1.94912), sum); // TODO rm
      tmp[m + threadIdx.x * BATCH_SIZE] = sum;
      // tmp[m * THREADS_PER_BLOCK + threadIdx.x] = sum;
#ifdef DEBUG
      cuCheck(sum);
#endif
    }
  }

  // TODO do smarter agg
  // sync all (incl non-aggregating cores)
  __syncthreads();

  // aggregate locally (within blocks)
  if (threadIdx.x == 0) {
    // for each y-datapoint in current batch
    for(unsigned int m = 0; m < BATCH_SIZE; ++m) {
      WTYPE_cuda sum;
      sum = ZERO;
      for (unsigned int k = 0; k < THREADS_PER_BLOCK; ++k)
        sum = hipCadd(sum, tmp[m + k * BATCH_SIZE]);
      // for (unsigned int k = 0; k < THREADS_PER_BLOCK; ++k)
      //   sum = hipCadd(sum, tmp[k + m * THREADS_PER_BLOCK]);

#ifdef DEBUG
      cuCheck(sum);
#endif

      // TODO foreach batch element
      // y[blockIdx.x + m * BLOCKDIM] = sum;
      const unsigned int i = blockIdx.x + m * BLOCKDIM;
      y[i] = sum.x;
      y[i + BLOCKDIM * BATCH_SIZE] = sum.y;
      // y[m + blockIdx.x * BATCH_SIZE] = sum;
    }
  }

  // do not sync blocks, exit kernel and agg block results locally or in diff kernel
}


__global__ void zip_arrays(double *__restrict__ a, double *__restrict__ b, size_t len, WTYPE_cuda *out) {
  // convert two arrays into array of tuples (i.e. complex numbers)
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < len; i+=stride) {
    out[i] = make_hipDoubleComplex(a[i], b[i]);
  }
}

template<typename Iterator, typename T, typename BinaryOperation, typename Pointer>
__global__ void reduce_kernel(Iterator first, Iterator last, T init, BinaryOperation binary_op, Pointer result)
{
  // from https://github.com/thrust/thrust/blob/master/examples/cuda/async_reduce.cu
  *result = thrust::reduce(thrust::cuda::par, first, last, init, binary_op);
}


__global__ void kernel1(WTYPE_cuda *x, STYPE *u, WTYPE_cuda  *y, STYPE *v)
{
  // Single kernel, used in y_i = \sum_j superposition_single(y_i,x_j)
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  WTYPE_cuda sum = ZERO;

  for(int n = 0; n < N; ++n)
    sum = hipCadd(superposition_single(n, i, x, u, v, 1), sum);

  y[i] = sum;
}

#endif
