#include "hip/hip_runtime.h"
#include "macros.h"

#ifdef DEBUG
#include <assert.h>
#endif
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <complex.h>
#include <hip/hip_complex.h>


inline
hipError_t cu(hipError_t result)
{
  // check for cuda errors
#if defined(DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %sn", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__device__ void cuCheck(hipDoubleComplex  z) {
  double a = hipCreal(z), b = hipCimag(z);
  if (isnan(a)) printf("cu found nan re\n");
  if (isinf(a)) printf("cu found inf re\n");
  if (isnan(b)) printf("cu found nan I\n");
  if (isinf(b)) printf("cu found inf I\n");
}

inline
__device__ double angle(hipDoubleComplex  z) {
  return atan2(hipCreal(z), hipCimag(z));
}

inline
__device__ hipDoubleComplex polar(double a, double phi) {
  // return the complex number a * exp(phi * imag)
  hipDoubleComplex res;
  sincos(phi, &res.x, &res.y);
  return hipCmul(make_hipDoubleComplex(a, 0), res);
}


__global__ void kernel_zero(WTYPE_cuda *x, size_t n) {
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < n; i += stride)
    x[i] = ZERO;
}

// TODO consider non-complex types (double real, double imag)
// and check computational cost
inline
__device__ WTYPE_cuda K(size_t i, size_t j, WTYPE_cuda *x, STYPE *u, STYPE *v, const char inverse) {
  // TODO unpack input to u1,u2,3 v1,v2,v3?
  // TODO consider unguarded functions, intrinsic functions
#ifdef DEBUG
  assert(inverse == -1 || inverse == 1);
#endif

  size_t n = i * DIMS,
         m = j * DIMS; // TODO use struct?
  // TODO use softeningSquared?
  double
    distance = norm3d(v[m] - u[n], v[m+1] - u[n+1], v[m+2] - u[n+2]),
    amp = hipCabs(x[i]),
    phase = angle(x[i]);

#ifdef DEBUG
  // printf("amp %0.4f, %f\n", amp, softeningSquared);
  // amp += softeningSquared;
  // char direction = 1; // must be -1 or +1
  // DIMS == 3
  if (j > 0) assert(m > 0);
  if (distance == 0) { printf("ERROR: distance must be nonzero"); asm("trap;"); }
  // if (amp > 0) printf(">0 \ti: %i, abs: %0.4f, dis: %0.3f\n", i, amp, distance);
  // // TODO check overflows
  if (isnan(amp)) printf("found nan\n");
  if (isinf(amp)) printf("found inf\n");
  if (isnan(distance)) printf("found nan\n");
  if (isinf(distance)) printf("found inf\n");
  // if (amp > 0) printf("amp = %0.5f > 0\n", amp);
  // if (distance > 0) printf("dis: %0.4f\n\n", distance);
  hipDoubleComplex res = polar(amp, phase);
  if (amp > 0) assert(hipCabs(res) > 0);
#endif

  // TODO __ddiv_rd, __dmul_ru
  return polar(amp / distance, phase - distance * inverse * TWO_PI_OVER_LAMBDA);
}

// TODO optimize memory / prevent Shared memory bank conflicts for x,u arrays
__global__ void kernel3(WTYPE_cuda *x, STYPE *u, WTYPE_cuda *out, STYPE *v, const size_t i_batch, const char inverse)
{
  /** First compute local sum, then do nested aggregation
   *
   * out[BATCH_SIZE * blockDim] = array with output per block
   * v[BATCH_SIZE * DIM] = locations of y-datapoints
   */
  //
  __shared__ WTYPE_cuda tmp[THREADS_PER_BLOCK * BATCH_SIZE];
  // TODO use cuda.y-stride? - note the double for loop - how much memory fits in an SM?
  // TODO switch y-loop and x-loop and let sum : [BATCH_SIZE]? assuming y-batch is in local memory
  // printf("idx %i -", threadIdx.x);
  {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    size_t j;
    WTYPE_cuda sum;
    // printf("i_batch: %i", i_batch);

    // for each y-datapoint in current batch
    for(unsigned int m = 0; m < BATCH_SIZE; ++m) {
      sum = ZERO;
      j = m + i_batch * BATCH_SIZE;
      // sum = make_hipDoubleComplex(2,2);
      // Usage of stride allows <<<1,1>>> kernel invocation
      // TODO use 4/8/16 aligned x-point data? - test performance diffs
      for (size_t i = idx; i < N; i += stride) {
        // printf("i: %i", i);
        sum = hipCadd(K(i, j, x, u, v, inverse), sum);
        // printf("i: %i, j: %j, x: %f \n", i,j,x[i]);

#ifdef DEBUG
        // assert(hipCabs(K(i, j, x, u, v, inverse)) > 0);
#endif
        // printf("idx %i -", threadIdx.x);
        // sum = make_hipDoubleComplex(2,2);
        // TODO sum is unchanged outside this scope
        // if (hipCabs(sum) > 0) printf("c- ");
        // if (hipCabs(sum) > 0) printf(".......\n\n");
      }
      // printf("_.");
      // if (hipCabs(sum) > 0) printf("e ");
       // printf("m:%i ", m);
      // printf("ix: %i \\\\", m * THREADS_PER_BLOCK + threadIdx.x);
      // assert(m * THREADS_PER_BLOCK + threadIdx.x < THREADS_PER_BLOCK * BATCH_SIZE);
      tmp[m + threadIdx.x * BATCH_SIZE] = sum;
      // tmp[m * THREADS_PER_BLOCK + threadIdx.x] = sum;
#ifdef DEBUG
      cuCheck(sum);
      // assert(hipCabs(sum) > 0);
#endif
    }
    // printf("thr \n");
  }

  // sync all (incl non-aggregating cores)
  // printf("idx0 %i -", threadIdx.x);
  __syncthreads();

  // aggregate locally (within blocks)
  // TODO multiple stages? if x % 2 == 0 -> ..
  // printf("idx1 %i -", threadIdx.x);
  if (threadIdx.x == 0) {
    // for each y-datapoint in current batch
    for(unsigned int m = 0; m < BATCH_SIZE; ++m) {
      WTYPE_cuda sum;
      sum = ZERO;
      for (unsigned int k = 0; k < THREADS_PER_BLOCK; ++k)
        sum = hipCadd(sum, tmp[m + k * BATCH_SIZE]);
      // for (unsigned int k = 0; k < THREADS_PER_BLOCK; ++k)
      //   sum = hipCadd(sum, tmp[m + k * THREADS_PER_BLOCK]);
      // for (unsigned int k = 0; k < THREADS_PER_BLOCK; ++k)
      //   sum = hipCadd(sum, tmp[k + m * THREADS_PER_BLOCK]);

#ifdef DEBUG
      cuCheck(sum);
      // assert(hipCabs(sum) > 0);
#endif

      // TODO foreach batch element
      // sum = make_hipDoubleComplex(2,0);
      // printf(" sum: %f - ", hipCabs(sum));
      // if (hipCabs(sum) > 0) printf("la");
      // out[blockIdx.x + m * BLOCKDIM] = sum;

      out[m + blockIdx.x * BATCH_SIZE] = sum;
    }
  }

  // __syncthreads();
  // if (threadIdx.x == 0 && blockIdx.x == 0) {
  //   for(unsigned int m = 0; m < BATCH_SIZE; ++m)
  //     for (unsigned int k = 0; k < BLOCKDIM; ++k)
  //       out[m + k * BATCH_SIZE] = make_hipDoubleComplex(2,2);
  // }

  // do not sync blocks, exit kernel and agg block results locally or in diff kernel
}

__global__ void kernel1(WTYPE_cuda *x, STYPE *u, WTYPE_cuda  *y, STYPE *v)
{
  // Single kernel, used in y_i = \sum_j K(y_i,x_j)
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  WTYPE_cuda sum = ZERO;

  for(int n = 0; n < N; ++n)
    sum = hipCadd(K(n, i, x, u, v, 1), sum);

  y[i] = sum;
}
