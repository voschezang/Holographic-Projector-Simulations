// #define _POSIX_C_SOURCE 199309L

#include <assert.h>
#include <hip/hip_complex.h>
#include <hiprand.h>
#include <float.h>
#include <limits.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <thrust/host_vector.h> // unused in this file but causes error if omitted
#include <iostream>

#include "macros.h"
#include "kernel.cu"
#include "init.h"
#include "util.h"
#include "functions.h"

/**
 * Input x,u is splitted over GPU cores/threads
 * Output y,v is streamed (send in batches).
 *
 * It is assumed that x,u all fit in GPU memory, but not necessarily in cache
 * Batches containing parts of y,v are send back to CPU immediately
 *
 * Naming convention
 * i,j,k = indices in flattened arrays
 * n,m = counters
 * N,M = sizes
 *
 * e.g. n = [0,..,N-1]
 */


int main() {
  printf("\nHyperparams:");
  printf("\n"); printf(" N: %4i^2 =%6i", N_sqrt, N);
  printf("\t"); printf("STREAM_BATCH_SIZE: \t%8i", STREAM_BATCH_SIZE);
  printf("\t"); printf("N_BATCHES: %8i", N_BATCHES);

  printf("\n"); printf(" GRIDDIM: %8i", GRIDDIM);
  printf("\t"); printf("BLOCKDIM: \t\t%8i", BLOCKDIM);
  printf("\t"); printf("E[tasks] = %0.3fk", GRIDDIM * BLOCKDIM * 1e-3);
  printf("\t"); printf("\tN/thread: %i", N_PER_THREAD);
  printf("\n"); printf(" N_STREAMS %3i \t\tSTREAM SIZE: %i (x3)", N_STREAMS, STREAM_SIZE);
  printf("\n"); printf("BATCHES_PER_STREAM (x STREAM_BATCH_SIZE = N): %i (x %i = %i)\n", BATCHES_PER_STREAM, STREAM_BATCH_SIZE, BATCHES_PER_STREAM * STREAM_BATCH_SIZE);
  printf("KERNELS_PER_BATCH %3i \t\tKERNEL BATCH SIZE: %i\n", KERNELS_PER_BATCH, KERNEL_BATCH_SIZE);
  // if (BATCHES_PER_STREAM < BATCH_SIZE)
  //   printf("BATCHES_PER_STREAM (%i) < BATCH_SIZE (%i)\n", BATCHES_PER_STREAM, BATCH_SIZE);

  printf("\n"); printf("Memory lb: %0.2f MB\n", memory_in_MB());
  {
    // auto n = double{BLOCKDIM * BATCH_SIZE};
    // auto m = double{n * sizeof(WTYPE) * 1e-3};
    double n = BLOCKDIM * STREAM_BATCH_SIZE;
    double m = n * sizeof(WTYPE) * 1e-3;
    printf("Shared data (per block) (tmp): %i , i.e. %0.3f kB\n", n, m);
  }
  check_params();
  struct timespec t0, t1, t2;
  clock_gettime(CLOCK_MONOTONIC, &t0);

  // TODO use cmd arg for x length
  auto
    X = std::vector<WTYPE>(1, {1.0});

  auto
    U = std::vector<STYPE>(X.size() * DIMS),
    V = std::vector<STYPE>(N * DIMS),
    W = std::vector<STYPE>(N * DIMS);

  init_planes(U, V, W);
  summarize_double('u', U);
  summarize_double('v', V);

  clock_gettime(CLOCK_MONOTONIC, &t1);
  printf("runtime init: \t%0.3f\n", dt(t0, t1));
  printf("loop\n");
  printf("--- --- ---   --- --- ---  --- --- --- \n");
  hipProfilerStart();
#ifdef Y_TRANSFORM
  // if X does not fit on GPU then do y += transform(x') for each subset x' in X
  auto Y = transform<Direction::Backward>(X, U, V);
#endif

#ifdef Z_TRANSFORM
  printf("\nSecond transform:\n");
  auto Z = transform<Direction::Forward>(Y, V, W);
#else
  auto Z = std::vector<WTYPE>(1);
#endif

  // end loop
  clock_gettime(CLOCK_MONOTONIC, &t2);
  hipProfilerStop();
  printf("done\n");
  printf("--- --- ---   --- --- ---  --- --- --- \n");
  double time = dt(t1, t2);
  printf("runtime init: \t%0.3f\n", time);

#ifdef Z_TRANSFORM
  printf("TFLOPS:   \t%0.5f \t (%i FLOP_PER_POINT)\n",  \
         flops(time), FLOP_PER_POINT);
  printf("Bandwidth: \t%0.5f MB/s (excl. shared memory)\n", bandwidth(time, 2, 0));
  printf("Bandwidth: \t%0.5f MB/s (incl. shared memory)\n", bandwidth(time, 2, 1));
#else
  printf("TFLOPS:   \t%0.5f \t (%i FLOP_PER_POINT)\n",  \
         2*flops(time), 2*FLOP_PER_POINT);
  printf("Bandwidth: \t%0.5f Mb/s (excl. shared memory)\n", bandwidth(time, 1, 0));
  printf("Bandwidth: \t%0.5f MB/s (incl. shared memory)\n", bandwidth(time, 1, 1));
#endif

  check_cvector(Y);
  check_cvector(Z);

#ifdef Y_TRANSFORM
  summarize_c('y', Y);
#endif
#ifdef Z_TRANSFORM
  summarize_c('z', Z);
#endif

  // write_arrays<FileType::TXT>(x,y,z, u,v,w, N);
  write_arrays<FileType::TXT>(X,Y,Z, U,V,W);
	return 0;
}
