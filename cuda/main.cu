#include "hip/hip_runtime.h"
// #define _POSIX_C_SOURCE 199309L

#include <assert.h>
#include <hip/hip_complex.h>
#include <hiprand.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <thrust/host_vector.h> // unused in this file but causes error if omitted
#include <thrust/system/cuda/experimental/pinned_allocator.h>

#include "macros.h"
#include "hyper_params.h"
#include "params.h"
#include "kernel.cu"
#include "util.h"
#include "init.h"
#include "input.h"
#include "functions.cu"

/**
 * Input x,u is splitted over GPU cores/threads
 * Output y,v is streamed (send in batches).
 *
 * It is assumed that x,u all fit in GPU memory, but not necessarily in cache
 * Batches containing parts of y,v are send back to CPU immediately
 *
 * Naming convention
 * i,j,k = indices in flattened arrays
 * n,m = counters
 * N,M = sizes
 *
 * e.g. n = [0,..,N-1]
 */

int main(int argc, char** argv) {
  /**
   * Note there is an effective minimum distance between projected points,
   * but that projecting point further apart requires a higher sampling density.
   *
   * Projecting more that ~20 points may result in a higher (and less random)
   * noise floor.
   */
  auto params = input::read_args(argc, argv);
  Setup<size_t>
    &n_planes = params.n_planes,
    &n_per_plane = params.n_per_plane;

  const auto transformation = PROJECT_PHASE ? Transformation::Full : Transformation::Amplitude;
  const bool add_reference = transformation == Transformation::Amplitude;
  // const bool add_reference = false;

  // TODO rename non-spatial xyz,uvw to setup.obj, setup.projector etc

  std::vector<SPACE>
    u (DIMS * n_per_plane.obj),
    v (DIMS * n_per_plane.projector),
    w (DIMS * n_per_plane.projection);

#ifdef READ_INPUT
  print("Reading input files");
  // overwrite x,u
  auto x = read_bytes<WAVE>(std::string{"../tmp/x_phasor.input"});
  auto u = read_bytes<SPACE>(std::string{"../tmp/x_pos.input"});
  params.n_per_plane.obj = x.size();
  assert(params.n_per_plane.obj == n_per_plane.obj);
  printf("Number of input datapoints/plane: %u\n", x.size());
  printf("Number of input datapoints/plane: %u\n", u.size() / DIMS);
  assert(x.size() == u.size() / DIMS);
  {
    // scale pos, assume pos was normalized
    for (size_t i = 0; i < x.size(); ++i) {
      u[i * DIMS + 0] *= object_width;
      u[i * DIMS + 1] *= object_width;
    }
  }
#else
  // TODO use cmd arg for x length
  const auto shape = Shape::DottedCircle;
  // const auto shape = Shape::Circle;
  auto x = std::vector<WAVE>(n_per_plane.obj, from_polar(1.0));
#endif

  const Geometry
    projector = init::geometry(n_per_plane.projector),
    projection = init::geometry(n_per_plane.projection);
  print_info(projector, n_planes, n_per_plane);

  struct timespec t0, t1, t2;
  auto dt = std::vector<double>(max(n_planes.projection, 1L));
  clock_gettime(CLOCK_MONOTONIC, &t0);

  const auto y_plane = Plane {width: PROJECTOR_WIDTH,
                              offset: {x: 0., y: 0., z: 0.},
                              aspect_ratio: params.aspect_ratio.projector,
                              randomize: params.randomize};
  init::plane(v, y_plane);

  summarize_double('v', v); // TODO edit in case aspect ratio != 1
  clock_gettime(CLOCK_MONOTONIC, &t1);
  printf("Runtime init: \t%0.3f\n", diff(t0, t1));
  hipProfilerStart();

  // change offset in first dim
  // note that x,y,z correspond to the spatial dims
  for (auto& i : range(n_planes.obj)) {
    printf("x plane #%i\n", i);

    // linear/geometric interpolation
    const double ratio = i == 0 ? i : i / ((double) n_planes.obj - 1.);
    Cartesian<double> obj_offset = {x: lerp(params.obj_offset.x, ratio),
                                    y: lerp(params.obj_offset.y, ratio),
                                    z: gerp(params.obj_offset.z, ratio)};

    const auto x_plane = Plane {width: lerp(params.obj_width, ratio),
                                offset: obj_offset,
                                aspect_ratio: 1.,
                                randomize: false};
    printf("x_plane: %i, width: %e\n", i, x_plane.width);
#ifndef READ_INPUT
    const double modulate = i / (double) n_planes.obj;
    init::sparse_plane(u, shape, x_plane.width, x_plane.offset, modulate);
#endif

    const auto x_suffix = std::to_string(i);
    write_arrays(x, u, "x" + x_suffix, "u" + x_suffix, x_plane, 0, 0);
    printf("--- --- ---   --- --- ---  --- --- --- \n");

    // The projector distribution is obtained by doing a single backwards transformation
    // TODO if x does not fit on GPU then do y += transform(x') for each subset x' in x

    // dt[0] will be overwritten
    auto y = time_transform<Direction::Backwards, false, add_reference>(x, u, v, projector, &t1, &t2, &dt[0], true);
    check_cvector(y);

    if (i == 0) summarize_c('y', y);
    write_arrays(y, v, "y" + x_suffix, "v" + x_suffix, y_plane, dt[0], flops(dt[0], x.size(), y.size()));

    // The projection distributions at various locations are obtained using forward transformations
    for (auto& j : range(n_planes.projection)) {
      // skip half of forward transformations when simulating for prototype
      // TODO allow to disable forward transformation
      if (transformation == Transformation::Amplitude && n_planes.obj >= 4 && j % 2 == 1) continue;
      printf(" z plane #%i\n", j);
      const auto ratio = j == 0 ? 0 : j / ((double) n_planes.projection - 1.);
      const double width = gerp(params.projection_width, ratio);
      obj_offset.z = gerp(params.projection_z_offset, ratio);
      const auto z_plane = Plane {width: width,
                                  offset: obj_offset,
                                  aspect_ratio: params.aspect_ratio.projection,
                                  randomize: params.randomize};

      printf("z_plane: %i, width: %e\n", j, z_plane.width);
      init::plane(w, z_plane);
      // init::plane(w, z_plane, {obj_offset.x, obj_offset.y, z_plane.z_offset});

      // TODO mv z outside loop to avoid unnecessary mallocs
      // auto z = std::vector<WAVE>(n.z);
      auto z = time_transform<Direction::Forwards>(y, v, w, projection, &t1, &t2, &dt[j]);
      check_cvector(z);
      if (i == 0 && j == 0) summarize_c('z', z);

      const auto z_suffix = x_suffix + "_" + std::to_string(j);
      // TODO write this async, next loop can start already
      write_arrays(z, w, "z" + z_suffix, "w" + z_suffix, z_plane, dt[j], flops(dt[j], y.size(), z.size()));
    }

    if (n_planes.projection)
      print_result(dt, y.size(), n_per_plane.projection);
  }
  printf("--- --- ---   --- --- ---  --- --- --- \n");
  printf("done\n");
  hipProfilerStop();
	return 0;
}
