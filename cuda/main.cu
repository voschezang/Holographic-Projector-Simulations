#include "hip/hip_runtime.h"
// #define _POSIX_C_SOURCE 199309L

#include <assert.h>
#include <hip/hip_complex.h>
#include <hiprand.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <thrust/host_vector.h> // unused in this file but causes error if omitted

#include "macros.h"
#include "hyper_params.h"
#include "kernel.cu"
#include "init.h"
#include "util.h"
#include "functions.h"

/**
 * Input x,u is splitted over GPU cores/threads
 * Output y,v is streamed (send in batches).
 *
 * It is assumed that x,u all fit in GPU memory, but not necessarily in cache
 * Batches containing parts of y,v are send back to CPU immediately
 *
 * Naming convention
 * i,j,k = indices in flattened arrays
 * n,m = counters
 * N,M = sizes
 *
 * e.g. n = [0,..,N-1]
 */


int main() {
  const struct {size_t x,y,z;} n = {x: 1,
                                    y: N_sqrt * N_sqrt,
                                    z: N_sqrt * N_sqrt};
  // TODO add cmd line args
  // TODO struct n_planes .x .y. z
  const size_t
    n_x_planes = 4,
    n_z_planes = 1;

  // const Transformation projector = Transformation::Full;
  const auto transformation = Transformation::Amplitude2;
  // const bool hd = false;
  const bool hd = true;
  const auto shape = Shape::DottedCircle;
  // const auto shape = Shape::Circle;

  Params params = init::params(Variable::Width, n_z_planes, hd);
  const Geometry p = init::geometry(n.y);
  const bool add_const_source = transformation == Transformation::Amplitude2;
  const double height = params.projector.width * (hd ? 1080. / 1920. : 1.);
  print_info(p, n.x, n.y, n.z);

  struct timespec t0, t1, t2;
  // auto dt = std::vector<double>(n_x_planes * n_z_planes); // TODO
  auto dt = std::vector<double>(n_z_planes);
  clock_gettime(CLOCK_MONOTONIC, &t0);

  // TODO use cmd arg for x length

  // TODO scale input intensity, e.g. 1/n, and also for distance: sqrt(p)/r^2
  // s.t. sum of irradiance/power/amp is 1
  // i.e. n A/d = 1
  // TODO make this optimization optional, as it introduces some error
  auto x = std::vector<WTYPE>(n.x, from_polar(1.0, 0.0));
  auto v = init::plane(n.y, params.projector);

  summarize_double('v', v); // TODO edit in case hd == true
  clock_gettime(CLOCK_MONOTONIC, &t1);
  printf("Runtime init: \t%0.3f\n", diff(t0, t1));
  hipProfilerStart();

  // change offset in first dim
  // note that x,z now correspond to the spatial dims
  auto rel_x_offsets = linspace(n_x_planes, 0.2, 0.8);
  auto rel_y_offsets = linspace(n_x_planes, 0.8, 0.2);
  auto z_offsets = geomspace(n_x_planes, 0.4, 0.4);
  for (auto& i : range(n_x_planes)) {
    printf("x plane #%i\n", i);
    params.projector.z_offset = z_offsets[i];
    const double
      x_offset = rel_x_offsets[i] * params.projector.width,
      y_offset = rel_y_offsets[i] * height,
      modulate = i / (double) n_x_planes;
    auto u = init::sparse_plane(x.size(), shape, params.input.width, x_offset, y_offset, modulate);
    const auto x_suffix = std::to_string(i);
    write_arrays<FileType::TXT>(x, u, "x" + x_suffix, "u" + x_suffix, params.input);
    printf("--- --- ---   --- --- ---  --- --- --- \n");

    // The projector distribution is obtained by doing a single backwards transformation
    // TODO if x does not fit on GPU then do y += transform(x') for each subset x' in x

    v = init::plane(n.y, params.projector);

    // dt[0] will be overwritten
    auto y = time_transform<Direction::Backwards, add_const_source>(x, u, v, p, &t1, &t2, &dt[0], true);
    check_cvector(y);

    if (i == 0)
      summarize_c('y', y);

    write_arrays<FileType::TXT>(y, v, "y" + x_suffix, "v" + x_suffix, params.projector);
    // square amp and rm phase after saving
    if (transformation == Transformation::Amplitude2)
      square_amp(y);

    // The projection distributions at various locations are obtained using forward transformations
    auto p = init::geometry(n.z);
    for (auto& j : range(params.projections.size())) {
      // skip half of forward transformations when simulating for prototype
      if (transformation == Transformation::Amplitude2 && n_x_planes >= 4 && j % 2 == 1) continue;
      printf(" z plane #%i\n", j);
      auto w = init::plane(n.z, params.projections[j], x_offset, y_offset);
      // TODO mv z outside loop to avoid unnecessary mallocs
      // auto z = std::vector<WTYPE>(n.z);
      auto z = time_transform<Direction::Forwards>(y, v, w, p, &t1, &t2, &dt[j]);
      check_cvector(z);
      if (i == 0 && j == 0)
        summarize_c('z', z);

      const auto z_suffix = x_suffix + "_" + std::to_string(j);
      // TODO write this async, next loop can start already
      write_arrays<FileType::TXT>(z, w, "z" + z_suffix, "w" + z_suffix, params.projections[j]);
    }

    print_result(dt, y.size(), n.z);
  }
  printf("--- --- ---   --- --- ---  --- --- --- \n");
  printf("done\n");
  hipProfilerStop();
	return 0;
}
