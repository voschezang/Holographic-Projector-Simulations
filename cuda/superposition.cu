#include "hip/hip_runtime.h"
#ifndef KERNEL_SUPERPOSITION
#define KERNEL_SUPERPOSITION

#include <assert.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_complex.h>
#include <hipcub/hipcub.hpp>

#include "macros.h"
#include "hyper_params.h"
#include "util.h"
#include "kernel.cu"

// TODO consider unguarded functions, intrinsic functions
// TODO use softeningSquared?


// a transformation from projector to projection is forwards, vice versa is backwards
enum class Direction {Forwards, Backwards};

// template<Direction dir>
// inline __device__ double value() {
//   // manual conversion because <type_traits> lib is not yet supported
//   if (dir == Direction::Forward) return double{1.0};
//   else return double{-1.0};
// }


///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////
namespace superposition {
///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////

template<const Direction direction>
inline __host__ __device__ WAVE phasor_displacement(const Polar x, const double *u, const double *v) {
  // inline __host__ __device__ WAVE phasor_displacement(const double a, const double phi, const SPACE *u, const SPACE *v) {
  /**
   * Compute the phasor displacement single source datapoint, for some target location `v \in R^3`
   * `amp / distance * exp(phi \pm distance * 2 * pi / lambda)`
   */
  // const auto distance  = NORM_3D(v[0] - u[0], v[1] - u[1], v[2] - u[2]);
  const double distance = NORM_3D(v[0] - u[0], v[1] - u[1], v[2] - u[2]);
#if DEBUG
  assert(distance > 1e-9);
#endif
  if (direction == Direction::Forwards)
    return from_polar(x.amp / distance, x.phase + distance * TWO_PI_OVER_LAMBDA);
  else
    return from_polar(x.amp / distance, x.phase - distance * TWO_PI_OVER_LAMBDA);
}

template<const Direction direction>
__global__ void phasor_displacement(const Polar x, const double *u, const double *v, WAVE *y) {
  // in place
  y[0] = phasor_displacement<direction>(x, u, v);
}

template<Direction direction, int blockDim_x, int blockDim_y, Algorithm algorithm, bool shared_memory = false>
__global__ void per_block(
#ifdef RANDOMIZE_SUPERPOSITION_INPUT
                          hiprandState *state, const unsigned int seed, const unsigned int i_stream,
                          const unsigned int bin_size, const unsigned int bins_per_thread,
                          // const unsigned int N, const unsigned int M, const unsigned int N_stride, // TODO use uint
#endif
                          const size_t N, const size_t M, const size_t N_stride, // TODO use uint
                          const Polar *__restrict__ x,
                          const double *__restrict__ u,
                          const double *__restrict__ v,
                          WAVE *__restrict__ y_global,
                          const bool append_result = false) {
  // Ideally N == width but in case of underutilized batches equality does not hold.
  // M_stride is omitted because it is always equal to M
#ifdef DEBUG
  assert(blockDim.x * blockDim.y * blockDim.z <= 1024); // max number of threads per block
#endif
  const dim3
    tid (blockIdx.x * blockDim.x + threadIdx.x,
         blockIdx.y * blockDim.y + threadIdx.y),
    gridSize (blockDim.x * gridDim.x,
              blockDim.y * gridDim.y);

#ifdef RANDOMIZE_SUPERPOSITION_INPUT
  // // reset state after every launch
  // TODO reset only once per transformation?

  const unsigned int global_tid = tid.x + tid.y * gridSize.x;
  const unsigned int i_state = global_tid + i_stream * gridSize.x * gridSize.y;
  const size_t stride_x = gridSize.x * bin_size;
  hiprandState state_local;
  if (N > gridSize.x)
    state_local = state[i_state];

#endif

  if (algorithm == Algorithm::Naive) {
    for (size_t n = tid.x; n < N; n += gridSize.x) {
      // Note that "caching" x[n] per outer loop happens automatically by the compiler
      for (size_t m = tid.y; m < M; m += gridSize.y) {
#ifndef TEST_CONST_PHASE
        const WAVE y = phasor_displacement<direction>(x[n], &u[n * DIMS], &v[m * DIMS]);
#else
        const WAVE y = from_polar(1., 0.);
#endif
        const size_t i = Yidx(n, m, N_stride, M);
        // TODO add bool to template and use: y[] = y + int(append) y
        if (append_result)
          {y_global[i].x += y.x; y_global[i].y += y.y;}
        else
          y_global[i] = y;
        assert(!isinf(hipCabs(y_global[i])));
        assert(!isnan(hipCabs(y_global[i])));
      } }
  }
  else {
    // TODO compare enums hipcub::BlockReduceAlgorithm: hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY hipcub::BLOCK_REDUCE_WARP_REDUCTIONS
    typedef hipcub::BlockReduce<double, blockDim_x, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY, 1, 1, 700> BlockReduce;
    // typedef hipcub::BlockReduce<double, blockDim_x> BlockReduce;
    __shared__ typename BlockReduce::TempStorage y_shared[shared_memory ? blockDim_y : 1];
    // __shared__ typename BlockReduce::TempStorage y_shared[blockDim_y];

    for (size_t m = tid.y; m < M; m += gridSize.y) {

      // TODO mv condition to start of func
      if (tid.x < N) { // TODO this should cause deadlocks during BlockReduce (for certain geometry)
        // TODO add subfunctions for profiler
        WAVE y {0,0};

        // ------------------------------------------------------------
#ifdef RANDOMIZE_SUPERPOSITION_INPUT
        if (N > gridSize.x) {
          // assert(stride_x * bins_per_thread == N);
          for (size_t i_bin = 0; i_bin < bins_per_thread; ++i_bin) {
            // for (size_t n = global_tid * ; n <  ++n) {
            const size_t n_offset = i_bin * stride_x;
            // TODO use hiprand_uniform4
            const size_t n = bin_size                       \
              ? n_offset + randint(&state_local, bin_size)
              : randint(&state_local, N);
            // ? n_offset + bin_size * (1 - hiprand_uniform(&state_local))
            if (bin_size)
              assert(n - n_offset <= bin_size);

            y = hipCadd(y, phasor_displacement<direction>(x[n], &u[n * DIMS], &v[m * DIMS]));
          }
        }
        else
#endif
          for (size_t n = tid.x; n < N; n += gridSize.x) {
          // double p[3] = {0,0,0}, w[3] = {1,2,3};
            y = hipCadd(y, phasor_displacement<direction>(x[n], &u[n * DIMS], &v[m * DIMS])); // 2.07954 TFLOPS
          // y = hipCadd(y, phasor_displacement<direction>(x[0], &u[0 * DIMS], &v[m * DIMS])); // 2.58397 TFLOPS
          // y = hipCadd(y, phasor_displacement<direction>(x[n], &u[n * DIMS], p)); // 2.07954 TFLOPS
          // y = hipCadd(y, phasor_displacement<direction>({1,2}, w, &v[m * DIMS])); 2.98763 TFLOPS
          // y = hipCadd(y, phasor_displacement<direction>({1,2}, w, p)); // ~3.3 TFLOPS
        }
        // ------------------------------------------------------------
#ifdef TEST_CONST_PHASE
        for (size_t n = tid.x; n < N; n += gridSize.x)
          y = hipCadd(y, from_polar(1., 0.));
#endif
        // alt // https://github.com/thrust/thrust/blob/master/examples/sum_rows.cu
        if (shared_memory) {
          // TODO don't save result to every thread, only thread 0
          // TODO what about unused thread in reduction? -> should cause deadlock
          // Real part .x
          y.x = BlockReduce(y_shared[threadIdx.y]).Sum(y.x);
          // TODO is a sync here required?
          // __syncthreads();
          // TODO mv first global mem acces here? -> hide memory latency
          // Imaginary part .y
          y.y = BlockReduce(y_shared[threadIdx.y]).Sum(y.y);
          // __syncthreads();

          if (threadIdx.x == 0) {
#ifdef TEST_CONST_PHASE
            assert(blockDim_y == blockDim.y);
            assert(y.x == blockDim_x);
            assert(y.y == 0.);
#endif
            const size_t i = Yidx(blockIdx.x, m, MIN(N_stride, gridDim.x), M);
            if (append_result)
              {y_global[i].x += y.x; y_global[i].y += y.y;}
            else
              y_global[i] = y;
          }
        }
        else {
          const size_t i = Yidx(tid.x, m, MIN(N_stride, gridSize.x), M);
          // printf("y[%i] or y[%i, %i]: amp = %e, \tangle = %e\n", i, tid.x, m, hipCabs(y), angle(y));
          if (append_result)
            {y_global[i].x += y.x; y_global[i].y += y.y;}
          else
            y_global[i] = y;
        }
      }
    }
  }
#ifdef RANDOMIZE_SUPERPOSITION_INPUT
  // update global state
  // TODO only when NOT resetting the state in between kernels
  if (N > gridSize.x)
    state[i_state] = state_local;
#endif
}

///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////
} // end namespace
///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////
#endif
