#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <hip/hip_runtime.h>

#include "macros.h"
#include "hyper_params.h"
#include "util.h"
#include "init.h"
#include "kernel.cu"
#include "superposition.cu"

// host superposition functions

template<typename T = double>
inline void cp_batch_data_to_device(const T *v, T *v_pinned, DeviceVector<T> d_v, hipStream_t stream) {
  // any host memory involved in async/overlapping data transfers must be page-locked
  for (size_t i = 0; i < d_v.size; ++i)
    v_pinned[i] = v[i];

  cu ( hipMemcpyAsync( d_v.data, v_pinned, d_v.size * sizeof(T),
                        hipMemcpyHostToDevice, stream ) );
}

template<typename T = WAVE>
inline void cp_batch_data_to_host(const T *d_y, T *y_pinned, const size_t len, hipStream_t stream) {
  // any host memory involved in async/overlapping data transfers must be page-locked
  cu ( hipMemcpyAsync( y_pinned, d_y, len * sizeof(T),
                        hipMemcpyDeviceToHost, stream ) );
}

#define SuperpositionPerBlock(blockDim_y) {                             \
    assert(blockDim_x * blockDim_y <= 1024);                            \
    superposition::per_block<direction, blockDim_x, blockDim_y, algorithm, shared_memory> \
      <<< gridDim, blockDim, 0, stream >>>                              \
      (N, M, d_x_ptr, d_u_ptr, d_v, d_y_tmp, append_result);          \
  }

#define SuperpositionPerBlockHelper(blockDim_x) {                       \
    superposition_per_block_helper<direction, blockDim_x, algorithm, shared_memory> \
      (gridDim, blockDim, stream,                                       \
       N, M, d_x_ptr, d_u_ptr, d_v, d_y_tmp, append_result);          \
  }

template<Direction direction, unsigned int blockDim_x, Algorithm algorithm, bool shared_memory>
inline void superposition_per_block_helper(const dim3 gridDim, const dim3 blockDim, hipStream_t stream,
                                           const size_t N, const size_t M,
                                           const WAVE *d_x_ptr, const SPACE *d_u_ptr, const SPACE *d_v,
                                           WAVE *d_y_tmp, const bool append_result)
// double *d_y_tmp_re, double *d_y_tmp_im)
{
  // unrolled for loop to allow constant blockDim
  // TODO add computation for shared memory size
  switch (blockDim.y) {
  case   1: SuperpositionPerBlock(  1) break;
  case   2: SuperpositionPerBlock(  2) break;
  // case   4: SuperpositionPerBlock(  4) break;
  // case   8: SuperpositionPerBlock(  8) break;
  // case  16: SuperpositionPerBlock( 16) break;
  // case  32: SuperpositionPerBlock( 32) break;
// #if KERNEL_SIZE >= 4
  case   4: SuperpositionPerBlock(  4) break;
// #endif
// #if KERNEL_SIZE >= 8
  case   8: SuperpositionPerBlock(  8) break;
// #endif
// #if KERNEL_SIZE >= 16
  case  16: SuperpositionPerBlock( 16) break;
// #endif
// #if KERNEL_SIZE >= 32
  case  32: SuperpositionPerBlock( 32) break;
// #endif
  case  64: SuperpositionPerBlock( 64) break;
  // case  64: { if (!shared_memory) SuperpositionPerBlock( 64) } break; // TODO
  // case 128: SuperpositionPerBlock(128) break;
  // case 256: SuperpositionPerBlock(256) break;
  // case 512: SuperpositionPerBlock(512) break;
  default: {fprintf(stderr, "BlockSize.y: %u not implemented\n", blockDim.y); exit(1);}
  }
}

template<Direction direction, Algorithm algorithm, bool shared_memory>
inline void superposition_per_block(const dim3 gridDim, const dim3 blockDim, hipStream_t stream,
                                    const size_t N, const size_t M,
                                    const WAVE *d_x_ptr, const SPACE *d_u_ptr, const SPACE *d_v,
                                    WAVE *d_y_tmp, const bool append_result)
  // double *d_y_tmp_re, double *d_y_tmp_im)
{
  // unrolled for loop to allow constant blockDim
  // Note that the max number of threads per block is 1024
  switch (blockDim.x) {
  case   1: SuperpositionPerBlockHelper(  1) break;
  case   2: SuperpositionPerBlockHelper(  2) break;
  case   4: SuperpositionPerBlockHelper(  4) break;
// #if KERNEL_SIZE <= 128
  case   8: SuperpositionPerBlockHelper(  8) break;
// #endif
// #if KERNEL_SIZE <= 64
  case  16: SuperpositionPerBlockHelper( 16) break;
// #endif
// #if KERNEL_SIZE <= 32
  case  32: SuperpositionPerBlockHelper( 32) break;
// #endif
// #if KERNEL_SIZE <= 16
  case  64: SuperpositionPerBlockHelper( 64) break;
// #endif
// #if KERNEL_SIZE <= 8
  // case 128: SuperpositionPerBlockHelper(128) break;
// #endif
// #if KERNEL_SIZE <= 4
  // case 256: SuperpositionPerBlockHelper(256) break;
// #endif
// #if KERNEL_SIZE <= 2
  // case 512: SuperpositionPerBlockHelper(512) break;
// #endif
  default: {fprintf(stderr, "BlockSize.x: %u not implemented\n", blockDim.x); exit(1);}
  }
}

template<bool add_constant = false>
void normalize_amp(std::vector<WAVE> &c, double to = 1., bool log_normalize = false) {
  double max_amp = 0;
  for (size_t i = 0; i < c.size(); ++i)
    max_amp = fmax(max_amp, hipCabs(c[i]));

  if (max_amp < 1e-6) {
    printf("WARNING, max_amp << 1\n");
    return;
  }
  max_amp /= to;

  // zero constant is equivalent to no constant and will be removed by compiler
  const auto constant = from_polar(add_constant ? 1.0 : 0.0, ARBITRARY_PHASE);
  if (add_constant)
    max_amp *= 2.;

  for (size_t i = 0; i < c.size(); ++i) {
    if (add_constant) {
      c[i].x = c[i].x / max_amp + constant.x / 2.;
      c[i].y = c[i].y / max_amp + constant.y / 2.;
    } else {
      c[i].x = c[i].x / max_amp;
      c[i].y = c[i].y / max_amp;
    }
  }

  if (log_normalize)
    for (size_t i = 0; i < c.size(); ++i) {
      if (c[i].x > 0) c[i].x = -log(c[i].x);
      if (c[i].y > 0) c[i].y = -log(c[i].y);
    }
}

void rm_phase(std::vector<WAVE> &c) {
  // Set phase to zero, note that `a * exp(0 I) == {a, 0}`
  for (size_t i = 0; i < c.size(); ++i)
    c[i] = {hipCabs(c[i]), 0.};
}


template<Direction direction, Algorithm algorithm = Algorithm::Naive, bool shared_memory = false>
inline std::vector<WAVE> transform(const std::vector<WAVE> &x,
                                   const std::vector<SPACE> &u,
                                   const std::vector<SPACE> &v,
                                   const Geometry& p) {
#ifdef TEST_CONST_PHASE
  const size_t N = p.n.x;
  const size_t M = p.n.y;
#endif
  // x = input or source data, y = output or target data
  if (algorithm == Algorithm::Naive) assert(!shared_memory);

  // derive size of matrix y_tmp
  size_t tmp_out_size = p.batch_size.x * p.batch_size.y;
  if (algorithm == Algorithm::Alt)
    if (shared_memory)
      tmp_out_size = MIN(p.batch_size.x, p.gridDim.x) * p.batch_size.y;
    else
      tmp_out_size = MIN(p.batch_size.x, p.gridSize.x) * p.batch_size.y;

  assert(tmp_out_size > 0);
  assert(u[2] != v[2]);
  // printf("batch out size %lu\n", tmp_out_size);
  // printf("gridSize: %u, %u\n", p.gridSize.x, p.gridSize.y);
  // printf("geometry new: <<< {%u, %u}, {%u, %u} >>>\n", p.gridDim.x, p.gridDim.y, p.blockDim.x, p.blockDim.y);

#ifdef DEBUG
  assert(std::any_of(x.begin(), x.end(), abs_of_is_positive));
  assert(x.size() >= 1);
#endif
  if (x.size() < p.gridSize.x)
    printf("Warning, suboptimal input size: %u < %u\n", x.size(), p.gridSize.x);

  // TODO duplicate stream batches to normal memory if too large
  auto y = std::vector<WAVE>(p.n.y);

  // Copy CPU data to GPU, don't use pinned (page-locked) memory for input data
  const thrust::device_vector<WAVE> d_x = x;
  const thrust::device_vector<SPACE> d_u = u;
  // cast to pointers to allow usage in non-thrust kernels
  const WAVE* d_x_ptr = thrust::raw_pointer_cast(&d_x[0]);
  const auto d_u_ptr = thrust::raw_pointer_cast(&d_u[0]);

  // malloc data using pinned memory for all batches before starting streams
  // TODO consider std::unique_ptr<>
  WAVE *y_pinned_ptr, *d_y_tmp_ptr;
  SPACE *v_pinned_ptr, *d_v_ptr;
  // TODO don't use pinned memory for d_y_
  auto d_y_tmp  = init::malloc_vectors<WAVE>(        &d_y_tmp_ptr,  p.n_streams, tmp_out_size);
  auto d_v      = init::malloc_matrix<SPACE>(        &d_v_ptr,      p.n_streams, p.batch_size.y * DIMS);
  auto v_pinned = init::pinned_malloc_vectors<SPACE>(&v_pinned_ptr, p.n_streams, p.batch_size.y * DIMS);
  auto y_pinned = init::pinned_malloc_vectors<WAVE>( &y_pinned_ptr, p.n_streams, p.batch_size.y);

  // TODO d_b is too large
  // const auto d_unit = thrust::device_vector<WAVE>(p.batch_size.y, {1., 0.}); // unit vector for blas
  const auto d_unit = thrust::device_vector<WAVE>(tmp_out_size / p.batch_size.y, {1., 0.}); // unit vector for blas
  const auto *d_b = thrust::raw_pointer_cast(d_unit.data());

  hipStream_t streams[p.n_streams];
  hipblasHandle_t handles[p.n_streams];
  for (auto& stream : streams)
    cu( hipStreamCreate(&stream) );

  for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
    cuB( hipblasCreate(&handles[i_stream]) );
    hipblasSetStream(handles[i_stream], streams[i_stream]);
  }

  for (size_t i = 0; i < p.n_batches.y; i+=p.n_streams) {
    // size_t batch_size_x = p.batch_size.x;
    for (size_t n = 0; n < p.n_batches.x; ++n) {
      // // each final x-batch may be under-used/occupied
      // if (n == p.n_batches.x - 1) batch_size_x = N - n * p.batch_size.x;
      if (i == 0) {
        // cp x batch data for all streams and sync
        // TODO
      }
      for (size_t i_stream = 0; i_stream < p.n_streams; ++i_stream) {
        const auto m = i + i_stream;
        if (m >= p.n_batches.y) break;
        if (p.n_batches.y > 10 && m % (int) (p.n_batches.y / 10) == 0 && n == 0)
          printf("\tbatch %0.3fk / %0.3fk\n", m * 1e-3, p.n_batches.y * 1e-3);

        if (n == 0)
          cp_batch_data_to_device<SPACE>(&v[m * p.batch_size.y * DIMS],
                                         v_pinned[i_stream], d_v[i_stream],
                                         streams[i_stream]);

        const bool append_result = n > 0;
        const size_t xu_offset = n * p.batch_size.x;
        superposition_per_block<direction, algorithm, shared_memory> \
          (p.gridDim, p.blockDim, streams[i_stream], p.batch_size.x, p.batch_size.y,
           d_x_ptr + xu_offset, d_u_ptr + xu_offset * DIMS,
           d_v[i_stream].data, d_y_tmp[i_stream], append_result);
      }
#ifdef TEST_CONST_PHASE
      for (size_t i_stream = 0; i_stream < p.n_streams; ++i_stream) {
        const auto m = i + i_stream;
        if (m >= p.n_batches.y) break;
        hipStreamSynchronize(streams[i_stream]);
        auto d = thrust::device_vector<WAVE> (d_y_tmp[i_stream], d_y_tmp[i_stream] + tmp_out_size);
        auto h = thrust::host_vector<WAVE> (d);
        auto x_per_batch = p.batch_size.x * p.batch_size.y / tmp_out_size;
        // printf("%lu \t %lu \t %lu\n", tmp_out_size, p.batch_size.x * p.batch_size.y, x_per_batch);
        assert(x_per_batch > 0);
        for (size_t j = 0; j < tmp_out_size; ++j)
          assert(hipCabs(h[j]) == (1. + n % p.n_batches.x) * x_per_batch);
        for (size_t j = 0; j < tmp_out_size; ++j)
          assert(hipCabs(h[j]) == (1. + n ) * x_per_batch);
        }
#endif
    } // end for n in [0,p.n_batches.x)

    for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
      const auto m = i + i_stream;
      if (m >= p.n_batches.y) break;
      kernel::sum_rows<false>(tmp_out_size / p.batch_size.y, p.batch_size.y,
                              handles[i_stream], d_y_tmp[i_stream], d_b, d_y_tmp[i_stream]);
      // TODO transform `re, im => a, phi ` (complex to polar)
      cp_batch_data_to_host<WAVE>(d_y_tmp[i_stream], y_pinned[i_stream],
                                  p.batch_size.y, streams[i_stream]);
#ifdef TEST_CONST_PHASE
      hipStreamSynchronize(streams[i_stream]);
      for (size_t j = 0; j < p.batch_size.y; ++j)
        assert(hipCabs(y_pinned[i_stream][j]) == N);
#endif
    }
    for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
      const auto m = i + i_stream;
      if (m >= p.n_batches.y) break;
      hipStreamSynchronize(streams[i_stream]);
      // TODO stage copy-phase of next batch before copy/sync?
      for (size_t j = 0; j < p.batch_size.y; ++j)
        y[j + m * p.batch_size.y] = y_pinned[i_stream][j];
    }
  }

  // sync all streams before returning
  hipDeviceSynchronize();
#ifdef TEST_CONST_PHASE
  for (size_t j = 0; j < M; ++j)
    assert(hipCabs(y[j]) == N);
#endif

#ifdef DEBUG
  printf("done, destroy streams\n");
#endif

  for (unsigned int i = 0; i < p.n_streams; ++i)
    hipStreamDestroy(streams[i]);

#ifdef DEBUG
  printf("free device memory\n");
#endif

  for (auto& handle : handles)
    cuB( hipblasDestroy(handle) );

  cu( hipFree(d_y_tmp_ptr ) );
  cu( hipFree(d_v_ptr       ) );
  cu( hipHostFree(v_pinned_ptr ) );
  cu( hipHostFree(y_pinned_ptr ) );

#ifdef DEBUG
  size_t len = min(100L, y.size());
  assert(std::any_of(y.begin(), y.begin() + len, abs_of_is_positive));
#endif
  return y;
}

/**
 * Time the transform operation over the full input.
 * Do a second transformation if add_reference is true.
 */
template<Direction direction, bool add_constant_wave = false, bool add_reference_wave = false>
std::vector<WAVE> time_transform(const std::vector<WAVE> &x,
                                 const std::vector<SPACE> &u,
                                 const std::vector<SPACE> &v,
                                 const Geometry& p,
                                 struct timespec *t1, struct timespec *t2, double *dt,
                                 bool verbose = false) {
  clock_gettime(CLOCK_MONOTONIC, t1);
  auto weights = std::vector<double> {1,
                                      add_constant_wave ? 1 : 0,
                                      add_reference_wave ? 1 : 0};
  normalize(weights);

  // for 512x512 planes, griddim 128x1, blockdim 64x16, 1 stream:
  // transform with custom agg: 25.617345 s
  // transform naive (Alt algo) with shared memory: 9.337457 s
  // (2.7 speedup)
  // for one-to-many input: speedup was at least ~10

  std::vector<WAVE> y;
  switch (p.algorithm) {
  case 1: y = transform<direction, Algorithm::Naive, false>(x, u, v, p); break;
  case 2: y = transform<direction, Algorithm::Alt, false>(x, u, v, p); break;
  case 3: y = transform<direction, Algorithm::Alt, true>(x, u, v, p); break;
  default: {fprintf(stderr, "algorithm is incorrect"); exit(1); }
  }

  const bool shared_memory = true;
  // const bool shared_memory = false;
  // // auto y = transform<direction, Algorithm::Naive, shared_memory>(x, u, v, p);
  // auto y = transform<direction, Algorithm::Alt, shared_memory>(x, u, v, p);
  // average of transformation and constant if any
  normalize_amp<add_constant_wave>(y, weights[0] + weights[1]);

  assert(!add_constant_wave);
  if (add_reference_wave) {
    /**
     * Add single far away light source behind the second (v) plane,
     * with arbitrary (but constant) phase
     * adding the planar wave should happen before squaring the amplitude
    */
    // TODO do this on CPU?
    const double z_offset = v[2] - DISTANCE_REFERENCE_WAVE; // assume v[:, 2] is constant
    printf("ref v[2]: %e\n", v[2]);
    // auto y_reference = transform<direction, Algorithm::Naive, shared_memory>({from_polar(1.)}, {{0.,0., z_offset}}, v, p);
    auto y_reference = transform<direction, Algorithm::Alt, shared_memory>({from_polar(1.)}, {{0.,0., z_offset}}, v, p);
    normalize_amp<false>(y_reference, weights[2]);
    // let full reference wave (amp+phase) interfere with original wave
    add_complex(y, y_reference);
    // reset phase of result, because the projector is limited
    for (size_t i = 0; i < y.size(); ++i)
      y[i] = from_polar(hipCabs(y[i]), angle(y_reference[i]));
  }

  clock_gettime(CLOCK_MONOTONIC, t2);
  *dt = diff(*t1, *t2);
  if (verbose)
    print_result(std::vector<double>{*dt}, x.size(), y.size());

  return y;
}
