#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "macros.h"
#include "hyper_params.h"
#include "util.h"
#include "init.h"
#include "kernel.cu"
#include "superposition.cu"

// host superposition functions

template<typename T = double>
inline void cp_batch_data_to_device(const T *v, T *v_pinned, DeviceVector<T> d_v, hipStream_t stream) {
  // any host memory involved in async/overlapping data transfers must be page-locked
  for (size_t i = 0; i < d_v.size; ++i)
    v_pinned[i] = v[i];

  cu ( hipMemcpyAsync( d_v.data, v_pinned, d_v.size * sizeof(T),
                        hipMemcpyHostToDevice, stream ) );
}

template<typename T = WAVE>
inline void cp_batch_data_to_host(const T *d_y, T *y_pinned, const size_t len, hipStream_t stream) {
  // any host memory involved in async/overlapping data transfers must be page-locked
  cu ( hipMemcpyAsync( y_pinned, d_y, len * sizeof(T),
                        hipMemcpyDeviceToHost, stream ) );
}

#define SuperpositionPerBlockNaive(blockDim_y) {                        \
    assert(blockDim_x * blockDim_y <= 1024);                            \
    superposition::per_block_naive<direction, blockDim_x, blockDim_y, algorithm, shared_memory> \
      <<< gridDim, blockDim, 0, stream >>>                              \
      (p, N, M, d_x_ptr, d_u_ptr, d_v,                                  \
       d_y_block);                                                      \
  }

#define SuperpositionPerBlockNaiveHelper(blockDim_x) {                  \
    superposition_per_block_naive_helper<direction, blockDim_x, algorithm, shared_memory> \
      (gridDim, blockDim, stream,                                       \
       p, N, M, d_x_ptr, d_u_ptr, d_v,                                  \
       d_y_block);                                                      \
  }

template<Direction direction, unsigned int blockDim_x, Algorithm algorithm, bool shared_memory>
inline void superposition_per_block_naive_helper(const dim3 gridDim, const dim3 blockDim, hipStream_t stream,
                                                 const Geometry& p, const size_t N, const size_t M,
                                                 const WAVE *d_x_ptr, const SPACE *d_u_ptr, const SPACE *d_v,
                                                 WAVE *d_y_block)
                                          // double *d_y_block_re, double *d_y_block_im)
{
  // unrolled for loop to allow constant blockDim
  // TODO add computation for shared memory size
  switch (blockDim.y) {
  case   1: SuperpositionPerBlockNaive(  1) break;
  case   2: SuperpositionPerBlockNaive(  2) break;
  // case   4: SuperpositionPerBlockNaive(  4) break;
  // case   8: SuperpositionPerBlockNaive(  8) break;
  // case  16: SuperpositionPerBlockNaive( 16) break;
  // case  32: SuperpositionPerBlockNaive( 32) break;
#if KERNEL_SIZE >= 4
  case   4: SuperpositionPerBlockNaive(  4) break;
#endif
#if KERNEL_SIZE >= 8
  case   8: SuperpositionPerBlockNaive(  8) break;
#endif
#if KERNEL_SIZE >= 16
  case  16: SuperpositionPerBlockNaive( 16) break;
#endif
#if KERNEL_SIZE >= 32
  case  32: SuperpositionPerBlockNaive( 32) break;
#endif
  // case  64: SuperpositionPerBlockNaive( 64) break;
  // case 128: SuperpositionPerBlockNaive(128) break;
  // case 256: SuperpositionPerBlockNaive(256) break;
  // case 512: SuperpositionPerBlockNaive(512) break;
  default: {printf("BlockSize.y: %u not implemented\n", blockDim.y); exit(1);}
  }
}

template<Direction direction, Algorithm algorithm, bool shared_memory>
inline void superposition_per_block_naive(const dim3 gridDim, const dim3 blockDim, hipStream_t stream,
                                          const Geometry& p, const size_t N, const size_t M,
                                          const WAVE *d_x_ptr, const SPACE *d_u_ptr, const SPACE *d_v,
                                          WAVE *d_y_block)
  // double *d_y_block_re, double *d_y_block_im)
{
  // unrolled for loop to allow constant blockDim
  // Note that the max number of threads per block is 1024
  switch (blockDim.x) {
  case   1: SuperpositionPerBlockNaiveHelper(  1) break;
  case   2: SuperpositionPerBlockNaiveHelper(  2) break;
  case   4: SuperpositionPerBlockNaiveHelper(  4) break;
#if KERNEL_SIZE <= 128
  case   8: SuperpositionPerBlockNaiveHelper(  8) break;
#endif
#if KERNEL_SIZE <= 64
  case  16: SuperpositionPerBlockNaiveHelper( 16) break;
#endif
#if KERNEL_SIZE <= 32
  case  32: SuperpositionPerBlockNaiveHelper( 32) break;
#endif
#if KERNEL_SIZE <= 16
  case  64: SuperpositionPerBlockNaiveHelper( 64) break;
#endif
#if KERNEL_SIZE <= 8
  case 128: SuperpositionPerBlockNaiveHelper(128) break;
#endif
#if KERNEL_SIZE <= 4
  case 256: SuperpositionPerBlockNaiveHelper(256) break;
#endif
#if KERNEL_SIZE <= 2
  case 512: SuperpositionPerBlockNaiveHelper(512) break;
#endif
  default: {printf("BlockSize.x: %u not implemented\n", blockDim.x); exit(1);}
  }
}

#define SuperpositionPerBlock(size) {                                   \
    superposition::per_block<direction, size><<< p.gridDim, p.blockSize, 0, stream >>> \
    (p, d_x, Nx, d_u, &d_v[k * DIMS], &d_y_block[j] );                  \
  }

template<Direction direction>
inline void partial_superposition_per_block(const Geometry& p, const size_t Nx,
                                            const WAVE *d_x, const SPACE *d_u, SPACE *d_v,
                                            hipStream_t stream, double *d_y_block)
{
  assert(p.blockSize <= 512); // not implemented
  for (unsigned int i = 0; i < p.batch_size; ++i) {
    const unsigned int j = i * p.gridDim * p.kernel_size; // * 2
    const unsigned int k = i * p.kernel_size;
    switch (p.blockSize) {
    case   1: SuperpositionPerBlock(  1) break;
    case   2: SuperpositionPerBlock(  2) break;
    case   4: SuperpositionPerBlock(  4) break;
    case   8: SuperpositionPerBlock(  8) break;
    case  16: SuperpositionPerBlock( 16) break;
    case  32: SuperpositionPerBlock( 32) break;
    case  64: SuperpositionPerBlock( 64) break;
    case 128: SuperpositionPerBlock(128) break;
    case 256: SuperpositionPerBlock(256) break;
    // case 512: SuperpositionPerBlock(512) break;
    default: printf("BlockSize incorrect\n");
    }
  }
}

template<bool transpose = false>
inline void sum_rows(const size_t width, const size_t n_rows, hipblasHandle_t handle,
                     WAVE *d_a, const WAVE *d_b,
                     WAVE *d_y, const WAVE beta = {0., 0.}) {
  /**
   * GEMV: GEneral Matrix Vector multiplication
   * y = alpha * op(A)x + beta y
   * Note, argument width = lda = stride of matrix
   * Note, cublasDgemw should be at least as fast as cublasCgemw because of data alignment
   * However, it may require an additional transpose of the the aggregated data
   */
  // TODO use y from previous y batch for 2D batch
  // TODO use hipblasCgemv?
  const WAVE alpha = {1.};
#ifdef TEST_CONST_PHASE2
  {
    hipDeviceSynchronize();
    size_t n = width * n_rows;
    // printf("n: %lu\n", n);
    thrust::device_vector<WAVE> d (d_a, d_a + n);
    thrust::host_vector<WAVE> h = d;
    for (size_t i = 0; i < n; ++i) {
      // printf("i: %lu, x: %f, y: %f\n", i, h[i].x, h[i].y);
      if (h[i].x - 1. > 1e-6 || h[i].y > 1e-6)
        printf("err: i: %lu, x: %f, y: %f\n", i, h[i].x, h[i].y);
      assert(h[i].x == 1.);
      assert(h[i].y == 0.);
    }
  }
#endif

  if (transpose)
    cuB( hipblasZgemv(handle, HIPBLAS_OP_T, width, n_rows, &alpha, d_a, width, d_b, 1, &beta, d_y, 1) );
  else
    cuB( hipblasZgemv(handle, HIPBLAS_OP_N, n_rows, width, &alpha, d_a, n_rows, d_b, 1, &beta, d_y, 1) );

#ifdef TEST_CONST_PHASE2
  {
    hipDeviceSynchronize();
    size_t n = n_rows;
    thrust::device_vector<WAVE> d (d_y, d_y + n);
    thrust::host_vector<WAVE> h = d;
    for (size_t i = 0; i < n; ++i) {
      assert(h[i].x == (double) width);
      assert(h[i].y == 0.);
    }
  }
#endif
}

inline void sum_rows_thrust(const size_t width, const size_t n_rows, hipStream_t stream,
                            double *d_x, double *d_y) {
  // launch 1x1 kernel in the specified selected stream, from which multiple thrust are called indirectly
  // auto ptr = thrust::device_ptr<double>(d_x);
  thrust::device_ptr<double>
    x_ptr (d_x),
    y_ptr (d_y);
  kernel::reduce_rows<<< 1,1,0, stream >>>(x_ptr, width, n_rows, 0.0, thrust::plus<double>(), y_ptr);
}

inline void agg_batch_blocks(const Geometry& p, hipStream_t stream,
                             DeviceVector<double> d_y_batch,
                             double *d_y_block) {
  // aggregate d_y_block and save to d_y_batch
  auto y1 = thrust::device_ptr<double>(&d_y_batch.data[0]);
  auto y2 = thrust::device_ptr<double>(&d_y_batch.data[d_y_batch.size / 2]);
  // TODO is a reduction call for each datapoint really necessary?
  for (unsigned int m = 0; m < p.n_per_batch; ++m) {
    // Assume two independent reductions are at least as fast as a large reduction.
    // I.e. no kernel overhead and better work distribution
    thrust::device_ptr<double> ptr(d_y_block + m * p.gridDim);

    // launch 1x1 kernels in selected streams, which calls thrust indirectly inside that stream
    kernel::reduce<<< 1,1,0, stream >>>(ptr, ptr + p.gridDim, 0.0, thrust::plus<double>(), &y1[m]);
    ptr += p.gridDim * p.n_per_batch;
    kernel::reduce<<< 1,1,0, stream >>>(ptr, ptr + p.gridDim, 0.0, thrust::plus<double>(), &y2[m]);
  }
}

inline void agg_batch_naive(const size_t half_n, WAVE *y, hipStream_t stream,
                            double *d_x, double *d_y) {
  // wrapper for thrust call using streams
  // TODO replace zip by `re, im => a, phi ` (complex to polar)
  kernel::zip_arrays<<< KERNEL_SIZE,1, 0, stream >>>(d_x, d_x + half_n, half_n, (WAVE*) d_y);
	cu( hipMemcpyAsync(y, d_y, half_n * sizeof(WAVE),
                      hipMemcpyDeviceToHost, stream ) );
}

inline void agg_batch(const Geometry& p, WAVE *y, hipStream_t stream,
                      WAVE *d_y_stream, double *d_y_batch) {
  // wrapper for thrust call using streams
  // TODO replace zip by `re, im => a, phi ` (complex to polar)
  kernel::zip_arrays<<< 1,1 >>>(d_y_batch, &d_y_batch[p.n_per_batch], p.n_per_batch, d_y_stream);
	cu( hipMemcpyAsync(y, d_y_stream, p.n_per_batch * sizeof(WAVE),
                      hipMemcpyDeviceToHost, stream ) );
}

template<bool add_constant = false>
void normalize_amp(std::vector<WAVE> &c, double to = 1., bool log_normalize = false) {
  double max_amp = 0;
  for (size_t i = 0; i < c.size(); ++i)
    max_amp = fmax(max_amp, hipCabs(c[i]));

  if (max_amp < 1e-6) {
    printf("WARNING, max_amp << 1\n");
    return;
  }
  max_amp /= to;

  // zero constant is equivalent to no constant and will be removed by compiler
  const auto constant = from_polar(add_constant ? 1.0 : 0.0, ARBITRARY_PHASE);
  if (add_constant)
    max_amp *= 2.;

  for (size_t i = 0; i < c.size(); ++i) {
    if (add_constant) {
      c[i].x = c[i].x / max_amp + constant.x / 2.;
      c[i].y = c[i].y / max_amp + constant.y / 2.;
    } else {
      c[i].x = c[i].x / max_amp;
      c[i].y = c[i].y / max_amp;
    }
  }

  if (log_normalize)
    for (size_t i = 0; i < c.size(); ++i) {
      if (c[i].x > 0) c[i].x = -log(c[i].x);
      if (c[i].y > 0) c[i].y = -log(c[i].y);
    }
}

void rm_phase(std::vector<WAVE> &c) {
  // Set phase to zero, note that `a * exp(0 I) == {a, 0}`
  for (size_t i = 0; i < c.size(); ++i)
    c[i] = {hipCabs(c[i]), 0.};
}


template<Direction direction, Algorithm algorithm = Algorithm::Naive, bool shared_memory = false>
inline std::vector<WAVE> transform_naive(const std::vector<WAVE> &x,
                                         const std::vector<SPACE> &u,
                                         const std::vector<SPACE> &v,
                                         const Geometry& p) {
  assert(u[2] != v[2]);
  const size_t N = u.size() / DIMS;
  const size_t M = v.size() / DIMS;
  const dim3
    gridDim (p.gridDim),
    blockDim (p.blockSize, KERNEL_SIZE),
    gridSize (blockDim.x * gridDim.x,
              blockDim.y * gridDim.y);

  // size_t gridSize = gridDim.x * gridDim.y * blockDim.x * blockDim.y;
  // Note that p.n_per_batch >= gridSize.y
  size_t batch_out_size = N * p.n_per_batch;
  if (algorithm == Algorithm::Alt)
    if (shared_memory)
      batch_out_size = MIN(N, gridDim.x) * p.n_per_batch; // TODO rename => kernel_out_size
    else
      batch_out_size = MIN(N, gridSize.x) * p.n_per_batch;

  printf("batch out size %lu\n", batch_out_size);
  printf("gridSize: %u, %u\n", gridSize.x, gridSize.y);
  printf("geometry new: <<< {%u, %u}, {%u, %u} >>>\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

#ifdef DEBUG
  assert(std::any_of(x.begin(), x.end(), abs_of_is_positive));
  assert(x.size() >= 1);
#endif
  if (x.size() < gridSize.x)
    printf("Warning, suboptimal input size: %u < %u\n", x.size(), gridSize.x);

  // TODO duplicate stream batches to normal memory if too large
  auto y = std::vector<WAVE>(M);

  // Copy CPU data to GPU, don't use pinned (page-locked) memory for input data
  const thrust::device_vector<WAVE> d_x = x;
  const thrust::device_vector<SPACE> d_u = u;
  // cast to pointers to allow usage in non-thrust kernels
  const auto d_x_ptr = thrust::raw_pointer_cast(&d_x[0]);
  const auto d_u_ptr = thrust::raw_pointer_cast(&d_u[0]);

  // malloc data using pinned memory for all batches before starting streams
  // TODO consider std::unique_ptr<>
  WAVE *y_pinned_ptr, *d_y_block_ptr;
  // double *d_y_batch_ptr;
  SPACE *v_pinned_ptr, *d_v_ptr;
  // TODO don't use pinned memory for d_y_
  auto d_y_block = init::malloc_vectors<WAVE>(&d_y_block_ptr, p.n_streams, batch_out_size);
  // auto d_y_batch = init::malloc_vectors<double>(&d_y_batch_ptr, p.n_streams, p.n_per_batch * 2);
  auto d_v       = init::malloc_matrix<SPACE>(&d_v_ptr, p.n_streams, p.n_per_batch * DIMS);
  auto v_pinned  = init::pinned_malloc_vectors<SPACE>(&v_pinned_ptr, p.n_streams, p.n_per_batch * DIMS);
  auto y_pinned  = init::pinned_malloc_vectors<WAVE>( &y_pinned_ptr, p.n_streams, p.n_per_batch);

  const auto d_unit = thrust::device_vector<WAVE>(batch_out_size, {1., 0.}); // unit vector for blas
  const WAVE *d_b = thrust::raw_pointer_cast(d_unit.data());

  hipStream_t streams[p.n_streams];
  hipblasHandle_t handles[p.n_streams];
  for (auto& stream : streams)
    cu( hipStreamCreate(&stream) );

  for (unsigned int i = 0; i < p.n_streams; ++i) {
      cuB( hipblasCreate(&handles[i]) );
      hipblasSetStream(handles[i], streams[i]);
  }

  for (size_t i = 0; i < p.n_batches; i+=p.n_streams) {
    // start each distinct kernel in batches

    for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
      const auto i_batch = i + i_stream;
      if (p.n_batches > 10 && i_batch % (int) (p.n_batches / 10) == 0)
        printf("\tbatch %0.3fk / %0.3fk\n", i_batch * 1e-3, p.n_batches * 1e-3);

      // TODO in case of 2D batches: only in case of new y-indices
      cp_batch_data_to_device<SPACE>(&v[i_batch * p.n_per_batch * DIMS],
                                     v_pinned[i_stream], d_v[i_stream],
                                     streams[i_stream]);

      superposition_per_block_naive<direction, algorithm, shared_memory>  \
        (gridDim, blockDim, streams[i_stream],
         p, N, p.n_per_batch, d_x_ptr, d_u_ptr, d_v[i_stream].data,
         d_y_block[i_stream]);
    }

    // do aggregations in separate stream-loops because of imperfect async functions calls on host
    // this may yield a ~2.5x speedup
    // TODO test again, with updated kernel funcs
    for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
      // TODO in case of 2D batches: save to d_y_batch, and then add to d_y_block

#ifdef TEST_CONST_PHASE2
      // hipStreamSynchronize(streams[i_stream]);
#endif
      sum_rows<false>(batch_out_size / p.n_per_batch, p.n_per_batch,
                      handles[i_stream], d_y_block[i_stream], d_b, d_y_block[i_stream]);
    }

    for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
      // const auto i_batch = i + i_stream;
      // re-use pinned memory
      // TODO transfrom `re, im => a, phi ` (complex to polar) (and immediately add to prev results)?
      // TODO in case of 2D batches: copy only if final batch for selected y-indices
      cp_batch_data_to_host<WAVE>(d_y_block[i_stream], y_pinned[i_stream],
                                  p.n_per_batch, streams[i_stream]);
    }

    // TODO stage copy-phase of next batch before copy?
    // or is it enough to sync a single stream at a time
    for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
      hipStreamSynchronize(streams[i_stream]);
      const auto i_batch = i + i_stream;
      for (size_t j = 0; j < p.n_per_batch; ++j)
        y[j + i_batch * p.n_per_batch] = y_pinned[i_stream][j];
    }
  }

  // sync all streams before returning
  hipDeviceSynchronize();

#ifdef DEBUG
  printf("done, destroy streams\n");
#endif

  for (unsigned int i = 0; i < p.n_streams; ++i)
    hipStreamDestroy(streams[i]);

#ifdef DEBUG
  printf("free device memory\n");
#endif

  for (auto& handle : handles)
    cuB( hipblasDestroy(handle) );

  cu( hipFree(d_y_block_ptr ) );
  // cu( hipFree(d_y_batch_ptr ) );
  cu( hipFree(d_v_ptr       ) );
  cu( hipHostFree(v_pinned_ptr ) );
  cu( hipHostFree(y_pinned_ptr ) );

#ifdef DEBUG
  size_t len = min(100L, y.size());
  assert(std::any_of(y.begin(), y.begin() + len, abs_of_is_positive));
#endif
  return y;
}


/**
 * d_x, d_u are stored in normal (non-pinned) GPU memory
 * d_y, d_v are stored partially, and copied back to CPU on the fly
 *
 * Additional temporary memory:
 * d_y_stream = reserved memory for each stream, containing batch result as complex doubles.
 * d_y_batch  = batch results, using doubles because thrust doesn't support cuComplexDouble
 * d_y_block  = block results (because blocks cannot sync), aggregated by thrust
 */
template<Direction direction>
inline std::vector<WAVE> transform(const std::vector<WAVE> &x,
                                    const std::vector<SPACE> &u,
                                    const std::vector<SPACE> &v,
                                    const Geometry& p) {
  assert(u[2] != v[2]);
  const size_t n = v.size() / DIMS;
#ifdef DEBUG
  assert(std::any_of(x.begin(), x.end(), abs_of_is_positive));
#endif
  if (x.size() < p.gridDim * p.blockSize)
    print("Warning, suboptimal input size");

  auto y = std::vector<WAVE>(n);

  // Copy CPU data to GPU, don't use pinned (page-locked) memory for input data
  const thrust::device_vector<WAVE> d_x = x;
  const thrust::device_vector<SPACE> d_u = u;
  // cast to pointers to allow usage in non-thrust kernels
  const auto d_x_ptr = thrust::raw_pointer_cast(&d_x[0]);
  const auto d_u_ptr = thrust::raw_pointer_cast(&d_u[0]);

  // Note that in case x.size < GRIDDIM the remaining entries in the agg array are zero
  hipStream_t streams[p.n_streams];
  // malloc data using pinned memory for all batches before starting streams
  // TODO consider std::unique_ptr<>
  WAVE *d_y_stream_ptr;
  double *d_y_block_ptr;
  double *d_y_batch_ptr;
  SPACE *d_v_ptr, *v_pinned_ptr;
  auto d_y_stream = init::pinned_malloc_vectors<WAVE>(&d_y_stream_ptr, p.n_streams, p.n_per_batch);
  auto d_y_block  = init::pinned_malloc_vectors<double>(&d_y_block_ptr, p.n_streams, 2 * p.n_per_batch * p.gridDim);
  auto d_y_batch  = init::pinned_malloc_matrix<double>(&d_y_batch_ptr, p.n_streams, 2 * p.n_per_batch);
  auto d_v        = init::pinned_malloc_matrix<SPACE>(&d_v_ptr, p.n_streams, p.n_per_batch * DIMS);
  auto v_pinned   = init::pinned_malloc_vectors<SPACE>(&v_pinned_ptr, p.n_streams, p.n_per_batch * DIMS);

  for (auto& stream : streams)
    hipStreamCreate(&stream);

  // assume n_batches is divisible by n_streams
  for (size_t i = 0; i < p.n_batches; i+=p.n_streams) {
    // start each distinct kernel in batches
    // TODO don't do this in case of non-uniform workloads

    for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
      const auto i_batch = i + i_stream;
      if (p.n_batches > 10 && i_batch % (int) (p.n_batches / 10) == 0)
        printf("\tbatch %0.3fk / %0.3fk\n", i_batch * 1e-3, p.n_batches * 1e-3);

      cp_batch_data_to_device<SPACE>(&v[i_batch * p.n_per_batch * DIMS],
                                     v_pinned[i_stream], d_v[i_stream], streams[i_stream]);

      partial_superposition_per_block<direction>(p, x.size(), d_x_ptr, d_u_ptr,
                                                 d_v[i_stream].data,
                                                 streams[i_stream], d_y_block[i_stream]);
    }

    // do aggregations in separate stream-loops because of imperfect async functions calls on host
    // this may yield a ~2.5x speedup
    for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
      agg_batch_blocks(p, streams[i_stream],
                       d_y_batch[i_stream],
                       d_y_block[i_stream]);
    }

    for (unsigned int i_stream = 0; i_stream < p.n_streams; ++i_stream) {
      const auto i_batch = i + i_stream;
      agg_batch(p, &y[i_batch * p.n_per_batch],
                streams[i_stream],
                d_y_stream[i_stream],
                d_y_batch[i_stream].data);
    }
  }

  // sync all streams before returning
  hipDeviceSynchronize();

#ifdef DEBUG
  printf("done, destroy streams\n");
#endif

  for (unsigned int i = 0; i < p.n_streams; ++i)
    hipStreamDestroy(streams[i]);

#ifdef DEBUG
  printf("free device memory\n");
#endif

  cu( hipHostFree(d_y_stream_ptr) );
  cu( hipHostFree(d_y_batch_ptr ) );
  cu( hipHostFree(d_y_block_ptr ) );
  cu( hipHostFree(d_v_ptr       ) );

  cu( hipHostFree(v_pinned_ptr ) );

#ifdef DEBUG
  size_t len = min(100L, y.size());
  assert(std::any_of(y.begin(), y.begin() + len, abs_of_is_positive));
#endif
  return y;
}

/**
 * Time the transform operation over the full input.
 * Do a second transformation if add_reference is true.
 */
template<Direction direction, bool add_constant_wave = false, bool add_reference_wave = false>
std::vector<WAVE> time_transform(const std::vector<WAVE> &x,
                                 const std::vector<SPACE> &u,
                                 const std::vector<SPACE> &v,
                                 const Geometry& p,
                                 struct timespec *t1, struct timespec *t2, double *dt,
                                 bool verbose = false) {
  clock_gettime(CLOCK_MONOTONIC, t1);
  auto weights = std::vector<double> {1,
                                      add_constant_wave ? 1 : 0,
                                      add_reference_wave ? 1 : 0};
  normalize(weights);

  // for 512x512 planes, griddim 128x1, blockdim 64x16, 1 stream:
  // transform with custom agg: 25.617345 s
  // transform naive (Alt algo) with shared memory: 9.337457 s
  // (2.7 speedup)
  // for one-to-many input: speedup was at least ~10

  // const bool shared_memory = true;
  const bool shared_memory = false;
  // auto y = transform<direction>(x, u, v, p);
  auto y = transform_naive<direction, Algorithm::Naive, shared_memory>(x, u, v, p);
  // auto y = transform_naive<direction, Algorithm::Alt, shared_memory>(x, u, v, p);
  // average of transformation and constant if any
  normalize_amp<add_constant_wave>(y, weights[0] + weights[1]);

  assert(!add_constant_wave);
  if (add_reference_wave) {
    /**
     * Add single far away light source behind the second (v) plane,
     * with arbitrary (but constant) phase
     * adding the planar wave should happen before squaring the amplitude
    */
    // TODO do this on CPU?
    const double z_offset = v[2] - DISTANCE_REFERENCE_WAVE; // assume v[:, 2] is constant
    printf("ref v[2]: %e\n", v[2]);
    auto y_reference = transform<Direction::Forwards>({from_polar(1.)}, {{0.,0., z_offset}}, v, p);
    normalize_amp<false>(y_reference, weights[2]);
    // let full reference wave (amp+phase) interfere with original wave
    add_complex(y, y_reference);
    // reset phase of result, because the projector is limited
    for (size_t i = 0; i < y.size(); ++i)
      y[i] = from_polar(hipCabs(y[i]), angle(y_reference[i]));
  }

  clock_gettime(CLOCK_MONOTONIC, t2);
  *dt = diff(*t1, *t2);
  if (verbose)
    print_result(std::vector<double>{*dt}, x.size(), y.size());

  return y;
}
