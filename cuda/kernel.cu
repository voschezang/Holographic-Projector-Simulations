#include "hip/hip_runtime.h"
#ifndef KERNEL
#define KERNEL

#include <assert.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_complex.h>

#include "macros.h"
#include "hyper_params.h"

#define cu(result) { cudaCheck((result), __FILE__, __LINE__); }


/** GPU version of std::vector
 */
template<typename T>
struct DeviceVector {
  T *data;
  size_t size;
};

inline
hipError_t cudaCheck(hipError_t result, const char *file, int line)
{
  // check for cuda errors
// #ifdef DEBUG
  if (result != hipSuccess) {
    fprintf(stderr, "[%s:%d] CUDA Runtime Error: %s\n", file, line, hipGetErrorString(result));
    // assert(result == hipSuccess);
    exit(result);
  }
// #endif
  return result;
}

inline
__host__ __device__ double norm3d_host(double a, double b, double c) {
  // simplified and host & device-compatible version of norm3d from CUDA math,  without overflow protection
  return pow(a * a + b * b + c * c, 0.5);
}

__host__ __device__ void cuCheck(hipDoubleComplex  z) {
  double a = hipCreal(z), b = hipCimag(z);
  if (isnan(a)) printf("cu found nan re\n");
  if (isinf(a)) printf("cu found inf re\n");
  if (isnan(b)) printf("cu found nan I\n");
  if (isinf(b)) printf("cu found inf I\n");
}

template <unsigned int blockSize, typename T>
inline __device__ void warp_reduce(volatile T *s, unsigned int i) {
  // example code from Nvidia
  if (blockSize >= 64) s[i] += s[i + 32];
  if (blockSize >= 32) s[i] += s[i + 16];
  if (blockSize >= 16) s[i] += s[i +  8];
  if (blockSize >=  8) s[i] += s[i +  4];
  if (blockSize >=  4) s[i] += s[i +  2];
  if (blockSize >=  2) s[i] += s[i +  1]; // TODO rm last line
}

// volatile WTYPE& operator=(volatile WTYPE&) volatile;

template <unsigned int blockSize>
inline __device__ void warp_reduce_complex(WTYPE *s, const unsigned int i) {
  // TODO assert size <= 2*WARP_SIZE
  // TODO if (1 < size <= 64) for (n = size / 2;;)
#pragma unroll
  for (int n = 32; n >= 1; n/=2) {
    if (blockSize >= n+n)
      s[i] = hipCadd(s[i], s[i + n]);

    __threadfence(); // TODO can this be moved inside the prev if
  }

  // // example code from Nvidia
  // if (size >= 64) s[i] = hipCadd(s[i], s[i + 32]);
  // __threadfence();
  // if (size >= 32) s[i] = hipCadd(s[i], s[i + 16]);
  // __threadfence();
  // if (size >= 16) s[i] = hipCadd(s[i], s[i +  8]);
  // __threadfence();
  // if (size >=  8) s[i] = hipCadd(s[i], s[i +  4]);
  // __threadfence();
  // if (size >=  4) s[i] = hipCadd(s[i], s[i +  2]);
  // __threadfence();
  // if (size >=  2) s[i] = hipCadd(s[i], s[i +  1]);
  // __threadfence();
}

inline __host__ __device__ void cos_sin(double x, double *cos, double *sin) {
  // Save cosine(x), sine(x) to &cos, &sin.
  // Flipped arguments for readability.
  sincos(x, sin, cos);
}

inline __host__ __device__ double angle(hipDoubleComplex c) {
  return atan2(c.y, c.x);
}

inline __host__ __device__ hipDoubleComplex from_polar(double r, double phi = 0.) {
  // Convert polar coordinates (r,phi) to Cartesian coordinates (re, im)
  // Using `r * e^(phi I) = r (cos phi + I sin phi)`
  // Note that result = {amp,0} if phase = 0, but adding such a branch may slow down performance
  hipDoubleComplex result;
  cos_sin(phi, &result.x, &result.y);
  return {r * result.x, r * result.y};
}

///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////
namespace kernel {
///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////

__global__ void zip_arrays(double *__restrict__ a, double *__restrict__ b, size_t len, WTYPE *out) {
  // convert two arrays into array of tuples (i.e. complex numbers)
  // i.e. transpose & flatten the matrix (a,b)
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < len; i+=stride) {
    out[i] = make_hipDoubleComplex(a[i], b[i]);
    // out[i] = {a[i], b[i]}; // TODO
  }
}

template<typename Iterator, typename T, typename BinaryOperation, typename Pointer>
__global__ void reduce(Iterator first, Iterator last, T init, BinaryOperation binary_op, Pointer result)
{
  // from https://github.com/thrust/thrust/blob/master/examples/cuda/async_reduce.cu
  *result = thrust::reduce(thrust::cuda::par, first, last, init, binary_op);
}

  ///////////////////////////////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////
} // end namespace
///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////
#endif
