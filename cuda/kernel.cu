#include "hip/hip_runtime.h"
#ifndef KERNEL
#define KERNEL

#include <assert.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "macros.h"
#include "hyper_params.h"

#define cu(result) cudaCheck((result), __FILE__, __LINE__)
#define cuB(result) cudaBlasCheck((result), __FILE__, __LINE__)


/** GPU version of std::vector
 */
template<typename T>
struct DeviceVector {
  T *data;
  size_t size;
};

inline
hipError_t cudaCheck(hipError_t result, const char *file, int line)
{
  // Note that max total blockSize is 1024
  // check for cuda errors
// #ifdef DEBUG
  if (result != hipSuccess) {
    fprintf(stderr, "[%s:%d] CUDA Runtime Error: %s\n", file, line, hipGetErrorString(result));
    // assert(result == hipSuccess);
    exit(result);
  }
// #endif
  return result;
}

inline
hipblasStatus_t cudaBlasCheck(hipblasStatus_t result, const char *file, int line)
{
  // check for cuda errors
  // #ifdef DEBUG
  if (result != HIPBLAS_STATUS_SUCCESS) {
    auto s = std::string();
    switch (result) {
      case HIPBLAS_STATUS_NOT_INITIALIZED:
        s = "HIPBLAS_STATUS_NOT_INITIALIZED";

      case HIPBLAS_STATUS_ALLOC_FAILED:
        s = "HIPBLAS_STATUS_ALLOC_FAILED";

      case HIPBLAS_STATUS_INVALID_VALUE:
        s = "HIPBLAS_STATUS_INVALID_VALUE";

      case HIPBLAS_STATUS_ARCH_MISMATCH:
        s = "HIPBLAS_STATUS_ARCH_MISMATCH";

      case HIPBLAS_STATUS_MAPPING_ERROR:
        s = "HIPBLAS_STATUS_MAPPING_ERROR";

      case HIPBLAS_STATUS_EXECUTION_FAILED:
        s = "HIPBLAS_STATUS_EXECUTION_FAILED";

      case HIPBLAS_STATUS_INTERNAL_ERROR:
        s = "HIPBLAS_STATUS_INTERNAL_ERROR";

      default:
        s = "<unknown>";
      }
    fprintf(stderr, "[%s:%d] cuBLAS Runtime Error: ", file, line);
    std::cout << s << '\n';
    exit(result);
  }
  // #endif
  return result;
}

inline
__host__ __device__ double norm3d_host(double a, double b, double c) {
  // simplified and host & device-compatible version of norm3d from CUDA math,  without overflow protection
  return pow(a * a + b * b + c * c, 0.5);
}

__host__ __device__ void cuCheck(hipDoubleComplex  z) {
  double a = hipCreal(z), b = hipCimag(z);
  if (isnan(a)) printf("cu found nan re\n");
  if (isinf(a)) printf("cu found inf re\n");
  if (isnan(b)) printf("cu found nan I\n");
  if (isinf(b)) printf("cu found inf I\n");
}

template <unsigned int blockSize, typename T>
inline __device__ void warp_reduce(volatile T *s, unsigned int i) {
  // example code from Nvidia
  if (blockSize >= 64) s[i] += s[i + 32];
  if (blockSize >= 32) s[i] += s[i + 16];
  if (blockSize >= 16) s[i] += s[i +  8];
  if (blockSize >=  8) s[i] += s[i +  4];
  if (blockSize >=  4) s[i] += s[i +  2];
  if (blockSize >=  2) s[i] += s[i +  1]; // TODO rm last line
}

template <unsigned int blockSize>
inline __device__ void warp_reduce_complex(WAVE *s, const unsigned int i) {
  // TODO assert size <= 2*WARP_SIZE
  // TODO if (1 < size <= 64) for (n = size / 2;;)
#pragma unroll
  for (int n = 32; n >= 1; n/=2) {
    if (blockSize >= n+n)
      s[i] = hipCadd(s[i], s[i + n]);

    __threadfence(); // TODO mv inside the prev if
  }

  // // example code from Nvidia
  // if (size >= 64) s[i] = hipCadd(s[i], s[i + 32]);
  // __threadfence();
  // if (size >= 32) s[i] = hipCadd(s[i], s[i + 16]);
  // __threadfence();
  // if (size >= 16) s[i] = hipCadd(s[i], s[i +  8]);
  // __threadfence();
  // if (size >=  8) s[i] = hipCadd(s[i], s[i +  4]);
  // __threadfence();
  // if (size >=  4) s[i] = hipCadd(s[i], s[i +  2]);
  // __threadfence();
  // if (size >=  2) s[i] = hipCadd(s[i], s[i +  1]);
  // __threadfence();
}


inline __host__ __device__ void cos_sin(const double x, double *cos, double *sin) {
  // Save cosine(x), sine(x) to &cos, &sin.
  // Flipped arguments for readability.
  sincos(x, sin, cos);
}

inline __host__ __device__ double angle(const hipDoubleComplex phasor) {
  return atan2(phasor.y, phasor.x);
}

inline __host__ __device__ hipDoubleComplex from_polar(const double r, const double phi = 0.) {
  // Convert polar coordinates (r,phi) to Cartesian coordinates (re, im)
  // Using `r * e^(phi I) = r (cos phi + I sin phi)`
  // TODO rename => to_phasor?
  // Note that result = {amp,0} if phase = 0, but adding such a branch may slow down performance
  hipDoubleComplex result;
  cos_sin(phi, &result.x, &result.y);
  return {r * result.x, r * result.y};
}

inline __host__ __device__ hipDoubleComplex to_polar(const hipDoubleComplex x) {
  return {hipCabs(x), angle(x)};
}

///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////
namespace kernel {
///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////

__global__ void zip_arrays(double *__restrict__ re, double *__restrict__ im, size_t len, hipDoubleComplex *__restrict__ out) {
  // convert two arrays into array of tuples (i.e. complex numbers)
  // i.e. transpose & flatten the matrix (a,b)
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;
  for (size_t i = idx; i < len; i+=stride)
    out[i] = {re[i], im[i]};
}

template<typename Iterator, typename T, typename BinaryOperation, typename Pointer>
__global__ void reduce(Iterator first, Iterator last, T init, BinaryOperation binary_op, Pointer result)
{
  // from https://github.com/thrust/thrust/blob/master/examples/cuda/async_reduce.cu
  *result = thrust::reduce(thrust::cuda::par, first, last, init, binary_op);
}

template<typename Iterator, typename T, typename BinaryOperation, typename Pointer>
__global__ void reduce_rows(Iterator first, const size_t width, const size_t n_rows, T init, BinaryOperation binary_op, Pointer results)
{
  // TODO use cuBlas gemv (with amortized plan)
  for (unsigned int i = 0; i < n_rows; ++i) {
    const size_t di = i * width;
    // from https://github.com/thrust/thrust/blob/master/examples/cuda/async_reduce.cu
    results[i] = thrust::reduce(thrust::cuda::par, first + di, first + di + width, init, binary_op);
  }
}


template<bool transpose = false>
inline void sum_rows(const size_t width, const size_t n_rows, hipblasHandle_t handle,
                     const WAVE *A, const WAVE *x,
                     WAVE *y, const WAVE beta = {0., 0.}) {
  /**
   * Sum all rows of matrix A. `A,x,y` must be device pointers.
   *
   * GEMV: GEneral Matrix Vector multiplication
   * `y = alpha * op(A)x + beta y`
   * Note, argument width = lda = stride of matrix
   */
  const WAVE alpha = {1., 0.};
// #ifdef TEST_CONST_PHASE
//   {
//     hipDeviceSynchronize();
//     size_t n = width * n_rows;
//     // printf("n: %lu\n", n);
//     thrust::device_vector<WAVE> d (A, A + n);
//     thrust::host_vector<WAVE> h = d;
//     for (size_t i = 0; i < n; ++i) {
//       // printf("i: %lu, x: %f, y: %f\n", i, h[i].x, h[i].y);
//       if (h[i].x - 1. > 1e-6 || h[i].y > 1e-6)
//         printf("err: i: %lu, x: %f, y: %f\n", i, h[i].x, h[i].y);
//       assert(h[i].x == 1.);
//       assert(h[i].y == 0.);
//     }
//   }
// #endif

  if (transpose)
    cuB( hipblasZgemv(handle, HIPBLAS_OP_T, width, n_rows, &alpha, A, width, x, 1, &beta, y, 1) );
  else
    cuB( hipblasZgemv(handle, HIPBLAS_OP_N, n_rows, width, &alpha, A, n_rows, x, 1, &beta, y, 1) );

// #ifdef TEST_CONST_PHASE
//   {
//     hipDeviceSynchronize();
//     size_t n = n_rows;
//     thrust::device_vector<WAVE> d (y, y + n);
//     thrust::host_vector<WAVE> h = d;
//     for (size_t i = 0; i < n; ++i) {
//       assert(h[i].x == (double) width);
//       assert(h[i].y == 0.);
//     }
//   }
// #endif
}

inline void sum_rows_thrust(const size_t width, const size_t n_rows, hipStream_t stream,
                            double *d_x, double *d_y) {
  // launch 1x1 kernel in the specified selected stream, from which multiple thrust are called indirectly
  // auto ptr = thrust::device_ptr<double>(d_x);
  thrust::device_ptr<double>
    x_ptr (d_x),
    y_ptr (d_y);
  kernel::reduce_rows<<< 1,1,0, stream >>>(x_ptr, width, n_rows, 0.0, thrust::plus<double>(), y_ptr);
}

  ///////////////////////////////////////////////////////////////////////////////////
  //////////////////////////////////////////////////////////////////////////////////
} // end namespace
///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////
#endif
