// #define _POSIX_C_SOURCE 199309L

#include <assert.h>
#include <hip/hip_complex.h>
#include <hiprand.h>
#include <float.h>
#include <limits.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <thrust/host_vector.h> // unused in this file but causes error if omitted

#include "macros.h"
#include "kernel.cu"
#include "init.h"
#include "util.h"
#include "functions.h"

/**
 * Input x,u is splitted over GPU cores/threads
 * Output y,v is streamed (send in batches).
 *
 * It is assumed that x,u all fit in GPU memory, but not necessarily in cache
 * Batches containing parts of y,v are send back to CPU immediately
 *
 * Naming convention
 * i,j,k = indices in flattened arrays
 * n,m = counters
 * N,M = sizes
 *
 * e.g. n = [0,..,N-1]
 */


int main() {
  printf("\nHyperparams:");
  printf("\n"); printf(" N: %4i^2 =%6i", N_sqrt, N);
  printf("\t"); printf("BATCH_SIZE:\t%8i", BATCH_SIZE);
  printf("\t"); printf("N_BATCHES: %8i", N_BATCHES);

  printf("\n"); printf(" GRIDDIM: %8i", GRIDDIM);
  printf("\t"); printf("BLOCKDIM: %8i", BLOCKDIM);
  printf("\t"); printf("E[tasks] = %0.3fk", GRIDDIM * BLOCKDIM * 1e-3);
  printf("\t"); printf("\tN/thread: %i", N_PER_THREAD);
  printf("\n"); printf(" N_STREAMS %3i \t\tSTREAM SIZE: %i (x3)", N_STREAMS, STREAM_SIZE);
  printf("\t"); printf("\tBATCHES_PER_STREAM (x BATCH_SIZE = N): %i (x %i = %i)\n", BATCHES_PER_STREAM, BATCH_SIZE, BATCHES_PER_STREAM * BATCH_SIZE);
  printf("KERNELS_PER_BATCH %3i \t\tKERNEL BATCH SIZE: %i\n", KERNELS_PER_BATCH, KERNEL_BATCH_SIZE);
  // if (BATCHES_PER_STREAM < BATCH_SIZE)
  //   printf("BATCHES_PER_STREAM (%i) < BATCH_SIZE (%i)\n", BATCHES_PER_STREAM, BATCH_SIZE);

  printf("\n"); printf("Memory lb: %0.2f MB\n", memory_in_MB());
  {
    double n = BLOCKDIM * BATCH_SIZE;
    double m = n * sizeof(WTYPE) * 1e-3;
    printf("Shared data (per block) (tmp): %i , i.e. %0.3f kB\n", n, m);
  }
  check_params();
  struct timespec t0, t1, t2;
	const size_t size = N * sizeof(WTYPE);
  clock_gettime(CLOCK_MONOTONIC, &t0);

  // host
  // vector<WTYPE> x1(0,N);
  WTYPE
    *x = (WTYPE *) malloc(size),
    *y = (WTYPE *) malloc(size),
    *z = (WTYPE *) malloc(size);

  STYPE
    *u = (STYPE *) malloc(DIMS * N * sizeof(STYPE)),
    *v = (STYPE *) malloc(DIMS * N * sizeof(STYPE)),
    *w = (STYPE *) malloc(DIMS * N * sizeof(STYPE));

  init_planes(x, u, v, w);
  summarize_double('u', u, N * DIMS);
  summarize_double('v', v, N * DIMS);

  clock_gettime(CLOCK_MONOTONIC, &t1);
  printf("runtime init: \t%0.3f\n", dt(t0, t1));
  printf("loop\n");
  printf("--- --- ---   --- --- ---  --- --- --- \n");
  hipProfilerStart();
  if (Y) {
    transform<Backward>(x, y, u, v);
  } else {
    printf("skipping y\n");
  }
  if (Z) {
    printf("\nSecond transform:\n");
    transform<Forward>(y, z, v, w);
    // transform(x, z, u, v, 1);
  }
  hipProfilerStop();

  // end loop
  clock_gettime(CLOCK_MONOTONIC, &t2);
  printf("done\n");
  printf("--- --- ---   --- --- ---  --- --- --- \n");
  double time = dt(t1, t2);
  printf("runtime init: \t%0.3f\n", time);

  if (Z) {
    printf("TFLOPS:   \t%0.5f \t (%i FLOP_PER_POINT)\n",  \
           flops(time), FLOP_PER_POINT);
    printf("Bandwidth: \t%0.5f MB/s (excl. shared memory)\n", bandwidth(time, 2, 0));
    printf("Bandwidth: \t%0.5f MB/s (incl. shared memory)\n", bandwidth(time, 2, 1));
  }
  else {
    printf("TFLOPS:   \t%0.5f \t (%i FLOP_PER_POINT)\n",  \
         2*flops(time), 2*FLOP_PER_POINT);
    printf("Bandwidth: \t%0.5f Mb/s (excl. shared memory)\n", bandwidth(time, 1, 0));
    printf("Bandwidth: \t%0.5f MB/s (incl. shared memory)\n", bandwidth(time, 1, 1));
  }
#ifdef DEBUG
  for (size_t i = 0; i < N2; ++i) {
    assert(hipCabs(y[i]) < DBL_MAX);
    assert(hipCabs(z[i]) < DBL_MAX);
  }
#endif

  if (Y) summarize_c('y', y, N);
  if (Z) summarize_c('z', z, N);

  printf("save results\n");
  write_arrays<TXT>(x,y,z, u,v,w, N);
  // write_arrays<GRID>(x,y,z, u,v,w, N);
  // write_arrays<DAT>(x,y,z, u,v,w, N);
  // write_arrays<DAT>(x,y,z, u,v,w, 100);
  printf("free xyz\n");
  free(x); free(y); free(z);
  printf("free uvw\n");
  free(u); free(v); free(w);
	return 0;
}
